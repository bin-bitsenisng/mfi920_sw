#include "hip/hip_runtime.h"
#/************************************************************************
 | Project Name: AFI910
 |    File Name: reshape.cu
 |  Description: detailed Description
 |-----------------------------------------------------------------------
 |               A U T H O R   I D E N T I T Y
 |-----------------------------------------------------------------------
 | Initials     Name                      Company                        
 | --------     ---------------------     -------------------------------
 | MHKIM        Myoungha Kim              bitsensing Corporation
 |-----------------------------------------------------------------------
 |              R E V I S I O N   H I S T O R Y
 |-----------------------------------------------------------------------
 | Date         Ver  Author  Description                                 
 | ----------  ----  ------  --------------------------------------------
 | 2021.03.12  0.0   mhkim   Creation;
 |***********************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>
#include <stdlib.h>
#include <time.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

#include "Sp_Parameters.h"

#define NUM_RX_PER_ARRAY     4
#define NUM_RX_ARRAY         4

#define NUM_RM_RX_PER_ARRAY  4
#define NUM_RM_RX_ARRAY      4

__global__ void lms_mode_reshape(int16_t* dst, int16_t* src)
{
    __shared__ uint16_t cache[SP_NUM_SAMPLES * NUM_RX_PER_ARRAY];
    __shared__ uint16_t reshaped_cache[SP_NUM_SAMPLES * NUM_RX_PER_ARRAY];
    int ti = threadIdx.x;
    int thread_size = blockDim.x;
    int input_tile_offset = NUM_RX_PER_ARRAY * SP_NUM_SAMPLES * (blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z);

    for(int i = 0; i < NUM_RX_PER_ARRAY; i++)
    {
        cache[i*thread_size + ti] = src[input_tile_offset + i*thread_size + ti];
    }

    __syncthreads();

    for(int i = 0; i < NUM_RX_PER_ARRAY; i++)
    {
        reshaped_cache[i*thread_size + ti] = cache[NUM_RX_PER_ARRAY * ti + i];
    }

    __syncthreads();

    for(int i = 0; i < NUM_RX_PER_ARRAY; i++)
    {
        dst[thread_size * (blockIdx.y + (gridDim.y * (3 - i)) + (gridDim.y * NUM_RX_PER_ARRAY * blockIdx.z) + (gridDim.y * NUM_RX_PER_ARRAY * gridDim.z * blockIdx.x)) + ti]\
        = reshaped_cache[i*thread_size + ti];
    }    
}

__global__ void ref_mode_reshape(int16_t* dst, int16_t* src)
{
    __shared__ uint16_t cache[SP_RM_NUM_SAMPLES * NUM_RM_RX_PER_ARRAY];
    __shared__ uint16_t reshaped_cache[SP_RM_NUM_SAMPLES * NUM_RM_RX_PER_ARRAY];
    int ti = threadIdx.x;
    int thread_size = blockDim.x;
    int input_tile_offset = NUM_RM_RX_PER_ARRAY * SP_RM_NUM_SAMPLES * (blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z);

    for(int i = 0; i < NUM_RM_RX_PER_ARRAY; i++)
    {
        cache[i*thread_size + ti] = src[input_tile_offset + i*thread_size + ti];
    }

    __syncthreads();

    for(int i = 0; i < NUM_RM_RX_PER_ARRAY; i++)
    {
        reshaped_cache[i*thread_size + ti] = cache[NUM_RX_PER_ARRAY * ti + i];
    }

    __syncthreads();

    for(int i = 0; i < NUM_RM_RX_PER_ARRAY; i++)
    {
        dst[thread_size * (blockIdx.y + (gridDim.y * (3 - i)) + (gridDim.y * NUM_RM_RX_PER_ARRAY * blockIdx.z) + (gridDim.y * NUM_RM_RX_PER_ARRAY * gridDim.z * blockIdx.x)) + ti]\
        = reshaped_cache[i*thread_size + ti];
    }    
}

void cudamalloc_data(int16_t* daq_rm_data, int16_t* daq_lr_data, int16_t* daq_mr_data, int16_t* daq_sr_data,   \
                       int16_t* rm_reshaped_data, int16_t* lr_reshaped_data, int16_t* mr_reshaped_data, int16_t* sr_reshaped_data)
{
    hipMallocManaged((void**)&daq_lr_data, sizeof(int16_t) * SP_NUM_SAMPLES * SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX);
    hipMallocManaged((void**)&daq_mr_data, sizeof(int16_t) * SP_NUM_SAMPLES * SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX);
    hipMallocManaged((void**)&daq_sr_data, sizeof(int16_t) * SP_NUM_SAMPLES * SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX);
    hipMallocManaged((void**)&daq_rm_data, sizeof(int16_t) * SP_RM_NUM_SAMPLES * SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX);   
}

void reshape_raw_data(int16_t* daq_data, int16_t* daq_reshaped_rm_data, int16_t* daq_reshaped_lr_data, int16_t* daq_reshaped_mr_data, int16_t* daq_reshaped_sr_data)
{
    struct timeval startTime, endTime;
    double diffTime;

    dim3 lms_mode_dimGrid(SP_NUM_TX, SP_NUM_CHIRPS, NUM_RX_ARRAY);
    dim3 ref_mode_dimGrid(SP_RM_NUM_TX, SP_RM_NUM_CHIRPS, NUM_RM_RX_ARRAY);

    gettimeofday(&startTime, NULL);

    ref_mode_reshape<<<ref_mode_dimGrid, SP_RM_NUM_SAMPLES>>>(daq_reshaped_rm_data, \
                                                                daq_data);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(daq_reshaped_lr_data, \
                                                                daq_data + SP_RM_NUM_SAMPLES * SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(daq_reshaped_mr_data, \
                                                                daq_data + SP_RM_NUM_SAMPLES * SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX + SP_NUM_SAMPLES * SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(daq_reshaped_sr_data, \
                                                                daq_data + SP_RM_NUM_SAMPLES * SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX + SP_NUM_SAMPLES * SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX * 2);

    hipStreamAttachMemAsync(NULL, daq_reshaped_rm_data, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(NULL, daq_reshaped_lr_data, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(NULL, daq_reshaped_mr_data, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(NULL, daq_reshaped_sr_data, 0, hipMemAttachHost);

    /*
    ref_mode_reshape<<<ref_mode_dimGrid, SP_RM_NUM_SAMPLES>>>(rm_reshaped_data, daq_rm_data);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(lr_reshaped_data, daq_lr_data);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(mr_reshaped_data, daq_mr_data);
    lms_mode_reshape<<<lms_mode_dimGrid, SP_NUM_SAMPLES>>>(sr_reshaped_data, daq_sr_data);
    
    hipStreamAttachMemAsync(NULL, rm_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, lr_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, mr_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, sr_reshaped_data, 0, hipMemAttachGlobal);
*/
	hipStreamSynchronize(NULL);

    gettimeofday(&endTime, NULL);
    diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
    //printf("gpu 5D reshape(): %lf s\n", diffTime);     
}

/*
int main(void)
{
    struct timeval startTime, endTime;
    double diffTime;

    uint16_t* lr_input_data;
    uint16_t* mr_input_data;
    uint16_t* sr_input_data;
    uint16_t* rm_input_data;
    hipMallocManaged((void**)&lr_input_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&mr_input_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&sr_input_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&rm_input_data, sizeof(uint16_t) * NUM_RM_RX_PER_ARRAY * NUM_RM_SAMPLES * NUM_RM_TX * NUM_RM_CHIRPS * NUM_RM_RX_ARRAY);

    uint16_t* lr_cpu_reshaped_data = (uint16_t*)malloc(sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    uint16_t* mr_cpu_reshaped_data = (uint16_t*)malloc(sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    uint16_t* sr_cpu_reshaped_data = (uint16_t*)malloc(sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    uint16_t* rm_cpu_reshaped_data = (uint16_t*)malloc(sizeof(uint16_t) * NUM_RM_RX_PER_ARRAY * NUM_RM_SAMPLES * NUM_RM_TX * NUM_RM_CHIRPS * NUM_RM_RX_ARRAY);
    
    uint16_t* lr_gpu_reshaped_data;
    uint16_t* mr_gpu_reshaped_data;
    uint16_t* sr_gpu_reshaped_data;
    uint16_t* rm_gpu_reshaped_data;
    hipMallocManaged((void**)&lr_gpu_reshaped_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&mr_gpu_reshaped_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&sr_gpu_reshaped_data, sizeof(uint16_t) * NUM_RX_PER_ARRAY * NUM_SAMPLES * NUM_TX * NUM_CHIRPS * NUM_RX_ARRAY);
    hipMallocManaged((void**)&rm_gpu_reshaped_data, sizeof(uint16_t) * NUM_RM_RX_PER_ARRAY * NUM_RM_SAMPLES * NUM_RM_TX * NUM_RM_CHIRPS * NUM_RM_RX_ARRAY);

    // 초기화
    srand(time(NULL));
    for(int i = 0; i < NUM_RX_ARRAY; i++)
    {
        for(int j = 0; j < NUM_CHIRPS; j++)
        {
            for(int k = 0; k < NUM_TX; k++)
            {
                for(int m = 0; m < NUM_SAMPLES; m++)
                {
                    for(int n = 0; n < NUM_RX_PER_ARRAY; n++)
                    {
                        lr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY +    \
                                        k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n] = rand() % 100;
                        mr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY +    \
                                        k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n] = rand() % 100;
                        sr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY +    \
                                        k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n] = rand() % 100;
                    }
                }
            }
        }
    }

    for(int i = 0; i < NUM_RM_RX_ARRAY; i++)
    {
        for(int j = 0; j < NUM_RM_CHIRPS; j++)
        {
            for(int k = 0; k < NUM_RM_TX; k++)
            {
                for(int m = 0; m < NUM_RM_SAMPLES; m++)
                {
                    for(int n = 0; n < NUM_RM_RX_PER_ARRAY; n++)
                    {
                        rm_input_data[i * NUM_RM_CHIRPS * NUM_RM_TX * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY + j * NUM_RM_TX * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY +    \
                                        k * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY + m * NUM_RM_RX_PER_ARRAY + n] = rand() % 100;
                    }
                }
            }
        }
    }

    gettimeofday(&startTime, NULL);

    for(int i = 0; i < NUM_RX_ARRAY; i++)
    {
        for(int j = 0; j < NUM_CHIRPS; j++)
        {
            for(int k = 0; k < NUM_TX; k++)
            {
                for(int m = 0; m < NUM_SAMPLES; m++)
                {
                    for(int n = 0; n < NUM_RX_PER_ARRAY; n++)
                    {
                        lr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]
                        = lr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n];
                        mr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]
                        = mr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n];
                        sr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]
                        = sr_input_data[i * NUM_CHIRPS * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + j * NUM_TX * NUM_SAMPLES * NUM_RX_PER_ARRAY + k * NUM_SAMPLES * NUM_RX_PER_ARRAY + m * NUM_RX_PER_ARRAY + n];
                    }
                }
            }
        }
    }

    for(int i = 0; i < NUM_RM_RX_ARRAY; i++)
    {
        for(int j = 0; j < NUM_RM_CHIRPS; j++)
        {
            for(int k = 0; k < NUM_RM_TX; k++)
            {
                for(int m = 0; m < NUM_RM_SAMPLES; m++)
                {
                    for(int n = 0; n < NUM_RM_RX_PER_ARRAY; n++)
                    {
                        rm_cpu_reshaped_data[k * NUM_RM_RX_ARRAY * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + i * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + n * NUM_RM_CHIRPS * NUM_RM_SAMPLES + j * NUM_RM_SAMPLES + m]
                        = rm_input_data[i * NUM_RM_CHIRPS * NUM_RM_TX * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY + j * NUM_RM_TX * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY + k * NUM_RM_SAMPLES * NUM_RM_RX_PER_ARRAY + m * NUM_RM_RX_PER_ARRAY + n];
                    }
                }
            }
        }
    }

    gettimeofday(&endTime, NULL);
    diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
    printf("cpu 5D reshape(): %lf s\n", diffTime);

    dim3 lms_mode_dimGrid(NUM_TX, NUM_CHIRPS, NUM_RX_ARRAY);
    dim3 ref_mode_dimGrid(NUM_RM_TX, NUM_RM_CHIRPS, NUM_RM_RX_ARRAY);

    gettimeofday(&startTime, NULL);

    lms_mode_reshape<<<lms_mode_dimGrid, NUM_SAMPLES>>>(lr_gpu_reshaped_data, lr_input_data);
    lms_mode_reshape<<<lms_mode_dimGrid, NUM_SAMPLES>>>(mr_gpu_reshaped_data, mr_input_data);
    lms_mode_reshape<<<lms_mode_dimGrid, NUM_SAMPLES>>>(sr_gpu_reshaped_data, sr_input_data);
    ref_mode_reshape<<<ref_mode_dimGrid, NUM_RM_SAMPLES>>>(rm_gpu_reshaped_data, rm_input_data);
    
    hipStreamAttachMemAsync(NULL, lr_gpu_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, mr_gpu_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, sr_gpu_reshaped_data, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, rm_gpu_reshaped_data, 0, hipMemAttachGlobal);
	hipStreamSynchronize(NULL);

    gettimeofday(&endTime, NULL);
    diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
    printf("gpu 5D reshape(): %lf s\n", diffTime);

    for(int k = 0; k < NUM_TX; k++)
    {
        for(int i = 0; i < NUM_RX_ARRAY; i++)
        {
            for(int n = 0; n < NUM_RX_PER_ARRAY; n++)
            {
                for(int j = 0; j < NUM_CHIRPS; j++)
                {
                    for(int m = 0; m < NUM_SAMPLES; m++)
                    {
                        if(lr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]\
                             != lr_gpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m] ||
                            mr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]\
                             != mr_gpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m] ||
                            sr_cpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m]\
                             != sr_gpu_reshaped_data[k * NUM_RX_ARRAY * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + i * NUM_RX_PER_ARRAY * NUM_CHIRPS * NUM_SAMPLES + n * NUM_CHIRPS * NUM_SAMPLES + j * NUM_SAMPLES + m])
                        {
                            //printf("gpu reshape failed.. [%d][%d][%d][%d][%d] cpu data: %d, gpu data: %d\n", k, i, n, j, m,
                            printf("lms_mode gpu reshape failed.. \n");
                            goto EXIT;
                        }
                    }
                }
            }
        }
    }

    for(int i = 0; i < NUM_RM_RX_ARRAY; i++)
    {
        for(int j = 0; j < NUM_RM_CHIRPS; j++)
        {
            for(int k = 0; k < NUM_RM_TX; k++)
            {
                for(int m = 0; m < NUM_RM_SAMPLES; m++)
                {
                    for(int n = 0; n < NUM_RM_RX_PER_ARRAY; n++)
                    {
                        if(rm_cpu_reshaped_data[k * NUM_RM_RX_ARRAY * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + i * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + n * NUM_RM_CHIRPS * NUM_RM_SAMPLES + j * NUM_RM_SAMPLES + m]\
                            != rm_gpu_reshaped_data[k * NUM_RM_RX_ARRAY * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + i * NUM_RM_RX_PER_ARRAY * NUM_RM_CHIRPS * NUM_RM_SAMPLES + n * NUM_RM_CHIRPS * NUM_RM_SAMPLES + j * NUM_RM_SAMPLES + m])
                        {
                           //printf("gpu reshape failed.. [%d][%d][%d][%d][%d] cpu data: %d, gpu data: %d\n", k, i, n, j, m,
                           printf("ref_mode gpu reshape failed.. \n");
                           goto EXIT;
                        }
                    }
                }
            }
        }
    }

EXIT:

    hipFree(lr_input_data);
    hipFree(mr_input_data);
    hipFree(sr_input_data);
    hipFree(rm_input_data);
    free(lr_cpu_reshaped_data);
    free(mr_cpu_reshaped_data);
    free(sr_cpu_reshaped_data);
    free(rm_cpu_reshaped_data);
    hipFree(lr_gpu_reshaped_data);
    hipFree(mr_gpu_reshaped_data);
    hipFree(sr_gpu_reshaped_data);
    hipFree(rm_gpu_reshaped_data);
}*/