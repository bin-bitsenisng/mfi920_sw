#include "hip/hip_runtime.h"
/************************************************************************
 | Project Name: AFI910
|    File Name: jetsonGPIO.h
|  Description: detailed Description
|-----------------------------------------------------------------------
|               A U T H O R   I D E N T I T Y
|-----------------------------------------------------------------------
| Initials     Name                      Company
| --------     ---------------------     -------------------------------
| sykang       KANG SOO YEON             Bitsensing
|-----------------------------------------------------------------------
|              R E V I S I O N   H I S T O R Y
|-----------------------------------------------------------------------
| Date         Ver  Author  Description
| ----------  ----  ------  --------------------------------------------
| 2020.10.15   0.0   sykang   Creation;
|***********************************************************************/

#include "host_dev.h"
#include "stdio.h"
#include "adi_dmdriver.h"
#include "adi_pmic_driver.h"
#include "bts_application.h"
#include <zmq.hpp>
#include <string.h>
#include <sys/select.h>
#include "reshape.cuh"
#include "timestamp.h"

#define CLEAR(x) memset(&(x), 0, sizeof(x))
#define REQ_BUFF_NUM 6
#define NUM_MIPI_CH 8
#define SPI_SPEED_FOR_PMIC 10000000
#define SPI_SPEED_FOR_RFIC 50000000


struct mipi_buffer_t
{
    void *start;
    size_t length;
};

//static enum io_method io_mode = IO_METHOD_MMAP;
static enum io_method io_mode = IO_METHOD_USERPTR;
struct mipi_buffer_t buffers[NUM_MIPI_CH][REQ_BUFF_NUM];
static unsigned int n_buffers;
static int mipi_ch_2_rfic_num[] = {1, 0, 3, 2,};
static unsigned int d_mode_len_byte_per_chip = D_MODE_NUM_SAMPLES_PER_RAMP * D_MODE_NUM_RAMPS * NUM_RX_PER_CHIP * 2; //*2 means that 1 sample = 2 bytes.
static unsigned int l_mode_len_byte_per_chip = L_MODE_NUM_SAMPLES_PER_RAMP * L_MODE_NUM_RAMPS * NUM_RX_PER_CHIP * 2; //*2 means that 1 sample = 2 bytes.


int  host_mipi_capt_stop(host_device_handle_t *host);
void host_mipi_close(host_device_handle_t *host);
void host_data_buf_init(host_device_handle_t *host);

extern zmq::socket_t daq_memcpy_socket;
extern zmq::message_t daq_memcpy_request;
daq_memcpy_info daq_memcpy_info_inst; 
void host_pheriph_init(host_device_handle_t *host){

   char temp_device_name[MAX_LENGTH] = "/dev/spidev0.0";

   printf(">>>> INIT HOST PROCESSOR <<<<\n");

   host->host_fe_pwr_en_pin   = gpio_id_CONN_FE_PWR_EN;
   host->host_rfic_nrst       = gpio_id_CONN_RFIC_nRST;
   host->host_rfic_npwdn      = gpio_id_CONN_RFIC_nPWDN;
   host->host_rfic_trig       = gpio_id_CONN_RFIC_TRIG;
   host->host_rfic_irq1       = gpio_id_CONN_FE_IRQ1;
   host->host_rfic_irq2       = gpio_id_CONN_FE_IRQ2;
   host->host_gpio0           = gpio_id_CONN_GPIO0;
   host->host_gpio1           = gpio_id_CONN_GPIO1;
   host->host_gpio2           = gpio_id_CONN_GPIO2;
   host->host_gpio3           = gpio_id_CONN_GPIO3;
   host->host_gpio4           = gpio_id_CONN_GPIO4;
   host->host_gpio5           = gpio_id_CONN_GPIO5;
   host->host_cpld_tms        = gpio_id_CONN_CPLD_TMS;
   host->host_cpld_tck        = gpio_id_CONN_CPLD_TCK;
   host->host_cpld_tdi        = gpio_id_CONN_CPLD_TDI;
   host->host_cpld_tdo        = gpio_id_CONN_CPLD_TDO;
   host->host_cpld_nclr       = gpio_id_CONN_CPLD_nCLR;
   host->host_cpld_oe         = gpio_id_CONN_CPLD_OE;
   host->host_led_red         = gpio_id_LED_RED;
   host->host_led_green       = gpio_id_LED_GREEN;
   host->host_led_blue        = gpio_id_LED_BLUE;


   strcpy(host->device_name, temp_device_name);
   host->spi_tr.tx_buf        = (unsigned long)host->mosi;
   host->spi_tr.rx_buf        = (unsigned long)host->miso;
   host->spi_tr.delay_usecs   = 0;
   host->spi_tr.bits_per_word = 8;
   host->spi_tr.speed_hz      = 10000000;
   host->handle_spi           = open(temp_device_name, O_RDWR);

   // set GPIOs
   gpioExport(host->host_fe_pwr_en_pin);
   gpioExport(host->host_rfic_nrst);
   gpioExport(host->host_rfic_npwdn);
   gpioExport(host->host_rfic_trig);
   gpioExport(host->host_rfic_irq1);
   gpioExport(host->host_rfic_irq2);
   gpioExport(host->host_gpio0);
   gpioExport(host->host_gpio1);
   gpioExport(host->host_gpio2); 
   gpioExport(host->host_gpio3); 
   gpioExport(host->host_gpio4);
   gpioExport(host->host_gpio5); 
   gpioExport(host->host_cpld_tms);
   gpioExport(host->host_cpld_tck);
   gpioExport(host->host_cpld_tdi);
   gpioExport(host->host_cpld_tdo);
   gpioExport(host->host_cpld_nclr);
   gpioExport(host->host_cpld_oe);
   gpioExport(host->host_led_red);
   gpioExport(host->host_led_green);
   gpioExport(host->host_led_blue);
   
   gpioSetDirection(host->host_fe_pwr_en_pin, outputPin);
   gpioSetDirection(host->host_rfic_nrst, outputPin);
   gpioSetDirection(host->host_rfic_npwdn, outputPin);
   gpioSetDirection(host->host_rfic_trig, outputPin);
   gpioSetDirection(host->host_rfic_irq1, inputPin);
   gpioSetDirection(host->host_rfic_irq2, inputPin);
   gpioSetDirection(host->host_gpio0, outputPin);
   gpioSetDirection(host->host_gpio1, outputPin);
   gpioSetDirection(host->host_gpio2, outputPin); 
   gpioSetDirection(host->host_gpio3, outputPin); 
   gpioSetDirection(host->host_gpio4, outputPin);
   gpioSetDirection(host->host_gpio5, outputPin); 
   gpioSetDirection(host->host_cpld_tms, outputPin);
   gpioSetDirection(host->host_cpld_tck, outputPin);
   gpioSetDirection(host->host_cpld_tdi, outputPin);
   gpioSetDirection(host->host_cpld_tdo, outputPin);
   gpioSetDirection(host->host_cpld_nclr, outputPin);
   gpioSetDirection(host->host_cpld_oe, outputPin);
   gpioSetDirection(host->host_led_red, outputPin);
   gpioSetDirection(host->host_led_green, outputPin);
   gpioSetDirection(host->host_led_blue, outputPin);

   gpioSetValue(host->host_led_blue, off);
   gpioSetValue(host->host_led_green, on);
   gpioSetValue(host->host_cpld_oe, on);
   gpioSetValue(host->host_cpld_nclr, on);

}

void host_spi_speed_change(host_device_handle_t *host){
   close(host->handle_spi);
   host->spi_tr.tx_buf        = (unsigned long)host->mosi;
   host->spi_tr.rx_buf        = (unsigned long)host->miso;
   host->spi_tr.delay_usecs   = 0;
   host->spi_tr.bits_per_word = 8;
   host->spi_tr.speed_hz      = 25000000;
   host->handle_spi           = open(host->device_name, O_RDWR);

}

void host_pheriph_deinit(host_device_handle_t *host){

   gpioSetValue(host->host_led_blue, on);
   gpioSetValue(host->host_led_green, off);

   gpioUnexport(host->host_fe_pwr_en_pin);
   gpioUnexport(host->host_rfic_nrst);
   gpioUnexport(host->host_rfic_npwdn);
   gpioUnexport(host->host_rfic_trig);
   gpioUnexport(host->host_rfic_irq1);
   gpioUnexport(host->host_rfic_irq2);
   gpioUnexport(host->host_gpio0);
   gpioUnexport(host->host_gpio1);
   gpioUnexport(host->host_gpio2); 
   gpioUnexport(host->host_gpio3); 
   gpioUnexport(host->host_gpio4);
   gpioUnexport(host->host_gpio5); 
   gpioUnexport(host->host_cpld_tms);
   gpioUnexport(host->host_cpld_tck);
   gpioUnexport(host->host_cpld_tdi);
   gpioUnexport(host->host_cpld_tdo);
   gpioUnexport(host->host_cpld_nclr);
   gpioUnexport(host->host_cpld_oe);
   gpioUnexport(host->host_led_red);
   gpioUnexport(host->host_led_green);
   gpioUnexport(host->host_led_blue);

   close(host->handle_spi);

   printf(">>>> DEINIT HOST PROCESSOR <<<<\n");

   host_mipi_capt_stop(host);
   host_mipi_close(host);
}

void host_rfic_power_up(host_device_handle_t *host){
   gpioSetValue(host->host_fe_pwr_en_pin, on);
   usleep(100000);
   gpioSetValue(host->host_rfic_npwdn, on);
   gpioSetValue(host->host_rfic_nrst, on);


   adi_pmic_PowerADAR690x(FIRST_PMIC);
   adi_pmic_PowerADAR690x(SECOND_PMIC);

}

void host_rfic_hard_reset(host_device_handle_t *host){     // rfic_nrst pin is associated with clk in CPLD...
   gpioSetValue(host->host_rfic_nrst, on);            // use soft reset using SPI.
   usleep(100000);
   gpioSetValue(host->host_rfic_nrst, off);
}

void print_spi_transaction(__u8 *miso, __u8 *mosi, __u32 length, __u8 slave)
{
	int i;
   printf("Select : %d,\n",slave);
	printf("MOSI : ");
	for (i = 0; i < length; i++)
		printf("%.2X ", mosi[i]);

   printf("\nMISO : ");
   for (i = 0; i < length; i++)
		printf("%.2X ", miso[i]);
   printf("\n");
}

void host_rfic_hw_trigger(host_device_handle_t *host){     // rfic_nrst pin is associated with clk in CPLD...
   gpioSetValue(host->host_rfic_trig, on);                 // use soft reset using SPI.
   usleep(10);
   gpioSetValue(host->host_rfic_trig, off);
}


static void errno_exit(const char *s)
{
    fprintf(stderr, "%s error %d, %s", s, errno, strerror(errno));
    printf("\n");
    exit(EXIT_FAILURE);
}

static int xioctl(int fh, int request, void *arg)
{
   int r;

   do
   {
      r = ioctl(fh, request, arg);
   } while (-1 == r && EINTR == errno);

   return r;
}


void host_mipi_open(host_device_handle_t *host)
{
   int ii;
   struct stat st;
   
   host->host_mipi_dev_name[0] = "/dev/video0";
   host->host_mipi_dev_name[1] = "/dev/video1";
   host->host_mipi_dev_name[2] = "/dev/video2";
   host->host_mipi_dev_name[3] = "/dev/video3";
   host->host_mipi_dev_name[4] = "/dev/video4";
   host->host_mipi_dev_name[5] = "/dev/video5";
   host->host_mipi_dev_name[6] = "/dev/video6";
   host->host_mipi_dev_name[7] = "/dev/video7";

   for (ii = HOST_MIPI_CH0_VCID0; ii < HOST_NUM_MIPI_CH; ii++)
   {
      if (-1 == stat(host->host_mipi_dev_name[ii], &st))
      {
         fprintf(stderr, "Cannot identify '%s': %d, %s\n", host->host_mipi_dev_name[ii], errno, strerror(errno));
         exit(EXIT_FAILURE);
      }

      if (!S_ISCHR(st.st_mode))
      {
         fprintf(stderr, "%s is no devicen", host->host_mipi_dev_name[ii]);
         exit(EXIT_FAILURE);
      }

      printf("open %s\n",host->host_mipi_dev_name[ii]);
      host->host_mipi_rfic_dev[ii] = open(host->host_mipi_dev_name[ii], O_RDWR /* required */ | O_NONBLOCK, 0);

      if (-1 == host->host_mipi_rfic_dev[ii])
      {
         fprintf(stderr, "Cannot open '%s': %d, %s\n", host->host_mipi_dev_name[ii], errno, strerror(errno));
         exit(EXIT_FAILURE);
      }
   }
   host_data_buf_init(host);   
}

void host_data_buf_init(host_device_handle_t *host)
{
   int total_len;
   int ii, jj;

   unsigned int page_size;

   total_len = (d_mode_len_byte_per_chip + 3*l_mode_len_byte_per_chip)*4;

   page_size = getpagesize ();
   total_len = (total_len + page_size - 1) & ~(page_size - 1);

   //host->data_buf[0] = (uint16_t *)calloc(total_len/2, sizeof(uint16_t));
   //host->data_buf[1] = (uint16_t *)calloc(total_len/2, sizeof(uint16_t));
   hipMallocManaged((void**)&host->data_buf[0], total_len/2 * sizeof(uint16_t), hipMemAttachHost);
   hipMallocManaged((void**)&host->data_buf[1], total_len/2 * sizeof(uint16_t), hipMemAttachHost);
   //hipMallocManaged((void**)&host->data_buf[0], total_len/2 * sizeof(uint16_t), hipMemAttachGlobal);
   //hipMallocManaged((void**)&host->data_buf[1], total_len/2 * sizeof(uint16_t), hipMemAttachGlobal);
   // host->data_buf = calloc(total_len/2, sizeof(uint16_t));
   for(ii = 0; ii < 4; ii++)
      host->sample_offset[ii] = mipi_ch_2_rfic_num[ii] * d_mode_len_byte_per_chip/2;
   for(jj = 0; jj < 3; jj++)
      for(ii = 0; ii < 4; ii++)
         host->sample_offset[4 + 4*jj + ii] = (d_mode_len_byte_per_chip/2*4) + (l_mode_len_byte_per_chip/2*4)*jj + mipi_ch_2_rfic_num[ii] * l_mode_len_byte_per_chip/2;
   for(ii=0;ii<16;ii++)
      printf("offset[%d] : %d\n", ii,  host->sample_offset[ii]);


}


static void host_init_mmap(host_device_handle_t *host, int num_device)
{
   struct v4l2_requestbuffers req;

   CLEAR(req);

   req.count = REQ_BUFF_NUM;
   req.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
   req.memory = V4L2_MEMORY_MMAP;

   if (-1 == xioctl(host->host_mipi_rfic_dev[num_device], VIDIOC_REQBUFS, &req))  //  Initiate Memory Mapping, User Pointer I/O or DMA buffer I/O
   {
      if (EINVAL == errno)
      {
         fprintf(stderr, "%s does not support memory mapping\n", host->host_mipi_dev_name[num_device]);
         exit(EXIT_FAILURE);
      }
      else
      {
         errno_exit("VIDIOC_REQBUFS");
      }
   }

   if (req.count < 2)
   {
      fprintf(stderr, "Insufficient buffer memory on %s\n", host->host_mipi_dev_name[num_device]);
      exit(EXIT_FAILURE);
   }

   for (n_buffers = 0; n_buffers < req.count; ++n_buffers)
   {
      struct v4l2_buffer buf;

      CLEAR(buf);

      buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
      buf.memory = V4L2_MEMORY_MMAP;
      buf.index = n_buffers;

      if (-1 == xioctl(host->host_mipi_rfic_dev[num_device], VIDIOC_QUERYBUF, &buf))
         errno_exit("VIDIOC_QUERYBUF");

      buffers[num_device][n_buffers].length = buf.length;
      buffers[num_device][n_buffers].start =
         mmap(NULL /* start anywhere */,
               buf.length,
               PROT_READ | PROT_WRITE /* required */,
               MAP_SHARED /* recommended */,
               host->host_mipi_rfic_dev[num_device], buf.m.offset);

      printf("device_num : %d, buff_ind : %d, buff_len : %d, buff_addr : 0x%X\n",num_device, buf.index, buf.length, buffers[num_device][n_buffers].start);

      if (MAP_FAILED == buffers[num_device][n_buffers].start)
         errno_exit("mmap");
   }
}

static void host_init_userptr(host_device_handle_t *host, int num_device)
{
   struct v4l2_requestbuffers req;

   CLEAR(req);

   //req.count = REQ_BUFF_NUM / 3;
   req.count = REQ_BUFF_NUM;
   req.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
   req.memory = V4L2_MEMORY_USERPTR;

   if (-1 == xioctl(host->host_mipi_rfic_dev[num_device], VIDIOC_REQBUFS, &req))
   {
      if (EINVAL == errno)
      {
         fprintf(stderr, "%s does not support user pointer i/o\n", host->host_mipi_dev_name[num_device]);
         exit(EXIT_FAILURE);
      }
      else
      {
         errno_exit("VIDIOC_REQBUFS");
      }
   }


}


void host_mipi_init(host_device_handle_t *host)
{
   struct v4l2_capability cap;
   struct v4l2_cropcap cropcap;
   struct v4l2_crop crop;
   struct v4l2_format fmt;
   unsigned int min;

   for(int ii = HOST_MIPI_CH0_VCID0; ii < HOST_NUM_MIPI_CH; ii++)
   {
      if (-1 == xioctl(host->host_mipi_rfic_dev[ii], VIDIOC_QUERYCAP, &cap))  // Query device capability
      {
         if (EINVAL == errno)
         {
            fprintf(stderr, "%s is no V4L2 device\n", host->host_mipi_dev_name[ii]);
            exit(EXIT_FAILURE);
         }
         else
         {
            errno_exit("VIDIOC_QUERYCAP");
         }
      }
      if (!(cap.capabilities & V4L2_CAP_VIDEO_CAPTURE))  // check video capture capability
      {
         fprintf(stderr, "%s is no video capture device\n", host->host_mipi_dev_name[ii]);
         exit(EXIT_FAILURE);
      }
      printf("open mipi device\n");
      /* Select video input, video standard and tune here. */

      CLEAR(cropcap);

      cropcap.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;

      if (0 == xioctl(host->host_mipi_rfic_dev[ii], VIDIOC_CROPCAP, &cropcap)) // Information about the video cropping and scaling abilities
      {
         crop.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
         crop.c = cropcap.defrect; /* reset to default */

         if (-1 == xioctl(host->host_mipi_rfic_dev[ii], VIDIOC_S_CROP, &crop)) //Get(VIDIOC_G_CROP) or set(VIDIOC_S_CROP) the current cropping rectangle
         {
            switch (errno)
            {
            case EINVAL:
                  /* Cropping not supported. */
                  break;
            default:
                  /* Errors ignored. */
                  break;
            }
         }
      }
      else
      {
         /* Errors ignored. */
      }

      CLEAR(fmt);

      fmt.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;

      fmt.fmt.pix.pixelformat = V4L2_PIX_FMT_Y12;
      // fmt.fmt.pix.field = V4L2_FIELD_INTERLACED;

      if (ii % 2 == 0) 
      {
         fmt.fmt.pix.width = D_MODE_NUM_SAMPLES_PER_RAMP*4;
         fmt.fmt.pix.height = D_MODE_NUM_RAMPS;
      }
      else
      {
         fmt.fmt.pix.width = L_MODE_NUM_SAMPLES_PER_RAMP*4;
         fmt.fmt.pix.height = L_MODE_NUM_RAMPS;
      }

      if (-1 == xioctl(host->host_mipi_rfic_dev[ii], VIDIOC_S_FMT, &fmt))  //Get(VIDIOC_G_FMT) or set(VIDIOC_S_FMT) the data format, try(VIDIOC_TRY_FMT) a format
         errno_exit("VIDIOC_S_FMT");


      fprintf(stderr, "****************************\n");
      fprintf(stderr, "radar_mode=%d \n", host->curr_profile);
      fprintf(stderr, "fmt.fmt.pix.width=%d \n", fmt.fmt.pix.width);
      fprintf(stderr, "fmt.fmt.pix.height=%d \n", fmt.fmt.pix.height);
      fprintf(stderr, "****************************\n");

      /* Buggy driver paranoia. */
      min = fmt.fmt.pix.width * 2;
      if (fmt.fmt.pix.bytesperline < min)
         fmt.fmt.pix.bytesperline = min;
      min = fmt.fmt.pix.bytesperline * fmt.fmt.pix.height;
      if (fmt.fmt.pix.sizeimage < min)
         fmt.fmt.pix.sizeimage = min;

      switch (io_mode)
      {
      case IO_METHOD_MMAP:
         host_init_mmap(host, ii);
         break;
      case IO_METHOD_USERPTR:
         host_init_userptr(host,ii);
         break;
      }
   }
}

void host_mipi_capt_start(host_device_handle_t *host)
{
   unsigned int i, cnt;
   enum v4l2_buf_type type;

   int jump_data_num_per_mode = 0;
   int jump_data_num_per_chip = 0;
  
   switch (io_mode)
   {
   case IO_METHOD_MMAP:
      for(int jj = HOST_MIPI_CH0_VCID0; jj < HOST_NUM_MIPI_CH; jj++)
      {
         for (i = 0; i < n_buffers; ++i)
         {
            struct v4l2_buffer buf;

            CLEAR(buf);
            buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
            buf.memory = V4L2_MEMORY_MMAP;
            buf.index = i;

            if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_QBUF, &buf))
                  errno_exit("VIDIOC_QBUF");
         }
         type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
         if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_STREAMON, &type))
            errno_exit("VIDIOC_STREAMON");
      }
      break;

   case IO_METHOD_USERPTR:
      cnt = 0;

      for(int jj = HOST_MIPI_CH0_VCID0; jj < HOST_NUM_MIPI_CH; jj++)
      {
         printf("userptr setting : %d\n",jj);
         if(jj % 2 == 0)    // D mode
         {
            jump_data_num_per_mode = 0;
            jump_data_num_per_chip = D_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP;

            for(int ii = 0; ii < REQ_BUFF_NUM; ii++){
               struct v4l2_buffer buf;
               CLEAR(buf);
               buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
               buf.memory = V4L2_MEMORY_USERPTR;
               buf.index = ii;
               buf.m.userptr = (unsigned long)(&host->data_buf[ii % 2][jump_data_num_per_mode + mipi_ch_2_rfic_num[jj / 2] * jump_data_num_per_chip]);
               //buf.m.userptr = (unsigned long)(host->data_buf + host->sample_offset[cnt]);
               //buf.m.userptr = (unsigned long)(host->data_buf[ii % 2]);
               buf.length = d_mode_len_byte_per_chip;

               if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_QBUF, &buf))
                  errno_exit("VIDIOC_QBUF");
            }
            cnt++;
         }
         if(jj % 2 == 1)    // LMS Mode
         {
            for(int ii = 0; ii < REQ_BUFF_NUM; ii++){
               if(ii % 3 == 0) // L mode
               {
                  jump_data_num_per_mode = D_MODE_TOTAL_NUM_SAMPLE*TOTAL_NUM_RX;
                  jump_data_num_per_chip = L_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP;
               }
               else if(ii % 3 == 1) // M mode
               {
                  jump_data_num_per_mode = (D_MODE_TOTAL_NUM_SAMPLE + L_MODE_TOTAL_NUM_SAMPLE) * TOTAL_NUM_RX;
                  jump_data_num_per_chip = M_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP;
               }
               else if(ii % 3 == 2) // S mode
               {
                  jump_data_num_per_mode = (D_MODE_TOTAL_NUM_SAMPLE + L_MODE_TOTAL_NUM_SAMPLE + M_MODE_TOTAL_NUM_SAMPLE) * TOTAL_NUM_RX;
                  jump_data_num_per_chip = S_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP;
               }

               struct v4l2_buffer buf;
               CLEAR(buf);
               buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
               buf.memory = V4L2_MEMORY_USERPTR;
               buf.index = ii;
               buf.m.userptr = (unsigned long)(&host->data_buf[ii / 3][jump_data_num_per_mode + mipi_ch_2_rfic_num[(jj - 1) / 2] * jump_data_num_per_chip]);
               //buf.m.userptr = (unsigned long)(host->data_buf[ii / 3]);
               buf.length = l_mode_len_byte_per_chip;

               if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_QBUF, &buf))
                  errno_exit("VIDIOC_QBUF");
            }
            cnt++;
         }
         type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
         if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_STREAMON, &type))
            errno_exit("VIDIOC_STREAMON");
      }
      break;

   }
}

static void write_files(int ch_num, const void *p, int size)
{
   int fd_buff = -1;
   int num;
   //Set file name
   time_t tm_time;
   struct tm *st_time;
   char filenanem[100];
   char filenamech[10];

   CLEAR(filenanem);

   time(&tm_time);
   st_time = localtime(&tm_time);
   strftime(filenanem, 100, "/mnt/ssd/%Y%m%d_%0l%M%S", st_time);
   // sprintf(filenamech, "_ch%d.raw",ch_num);
   // strcat(filenanem,filenamech);

   //fprintf(stderr, "%s\n", filenanem);

   fd_buff = open(filenanem, O_CREAT| O_WRONLY | O_TRUNC, 0644);
   if (fd_buff == -1)
   {
      perror("Open");
      exit(1);
   }

   if (write(fd_buff, p, size) != size)
   {
      perror("Write");
   }

   close(fd_buff);
}

static void process_image(int ch_num, const void *p, int size)
{
   // int out_buf;
   // if (out_buf)
   //    fwrite(p, size, 1, stdout);

   fflush(stderr);
   fprintf(stderr, ". size=%d \n", size);
   write_files(ch_num, p, size);
   fflush(stdout);
}

static int host_read_frame(host_device_handle_t *host)
{
   struct v4l2_buffer buf;
   unsigned int i;
   int ret;
   static int jump_data_num_per_mode;
   static int jump_data_num_per_chip;
   uint8_t b_num;

   b_num = host->buf_num;

   // jump_data_num = 0;
   // printf("host->curr_profile in host_read_frame : %d\n",host->curr_profile);

   switch(host->curr_profile)
   {
      case(D_MODE_BURST_NUM):
      {
         jump_data_num_per_mode = 0;
         jump_data_num_per_chip = D_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP*2; // *2 for 2byte data
         //memset(host->data_buf[b_num], 0, d_mode_len_byte_per_chip + l_mode_len_byte_per_chip * 3);
         break;
      }
      case(L_MODE_BURST_NUM):
      {
         jump_data_num_per_mode += D_MODE_TOTAL_NUM_SAMPLE*TOTAL_NUM_RX*2;
         jump_data_num_per_chip =  L_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP*2;
         break;
      }
      case(M_MODE_BURST_NUM):
      {
         jump_data_num_per_mode += L_MODE_TOTAL_NUM_SAMPLE*TOTAL_NUM_RX*2;
         jump_data_num_per_chip =  M_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP*2;
         break;
      }
      case(S_MODE_BURST_NUM):
      {
         jump_data_num_per_mode += M_MODE_TOTAL_NUM_SAMPLE*TOTAL_NUM_RX*2;
         jump_data_num_per_chip =  S_MODE_TOTAL_NUM_SAMPLE*NUM_RX_PER_CHIP*2;
         break;
      }
   }

   switch (io_mode)
   {
   case IO_METHOD_MMAP:

      CLEAR(buf);

      buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
      buf.memory = V4L2_MEMORY_MMAP;
      for(int ii = 0; ii < 4; ii++){
         if(host->curr_profile ==0)
            ret = xioctl(host->host_mipi_rfic_dev[2*ii], VIDIOC_DQBUF, &buf);
         else
            ret = xioctl(host->host_mipi_rfic_dev[2*ii+1], VIDIOC_DQBUF, &buf);

         if (-1 == ret)   // Exchange a buffer with the driver
         {
            switch (errno)
            {
            case EAGAIN:
                  return 0;
            case EIO:
                  /* Could ignore EIO, see spec. */

                  /* fall through */

            default:
                  errno_exit("VIDIOC_DQBUF");
            }
         }
         // assert(buf.index < n_buffers);
         
         /* copy from temp buffer to main data buffer */
         /*
         if(host->curr_profile == 0)
         {
            //memcpy((host->data_buf[b_num]) + (jump_data_num_per_mode + mipi_ch_2_rfic_num[ii] * jump_data_num_per_chip)/2, buffers[2*ii][buf.index].start, buf.bytesused);
            daq_memcpy_info_inst.dst = (void*)((host->data_buf[b_num]) + (jump_data_num_per_mode + mipi_ch_2_rfic_num[ii] * jump_data_num_per_chip)/2);
            daq_memcpy_info_inst.src = buffers[2*ii][buf.index].start;
            daq_memcpy_info_inst.copy_size = buf.bytesused;
            daq_memcpy_socket.send(&daq_memcpy_info_inst, sizeof(daq_memcpy_info_inst));
            //printf("[SEND] dst: 0x%x, src: 0x%x, size: %d\n", (uint32_t*)daq_memcpy_info_inst.dst, (uint32_t*)daq_memcpy_info_inst.src, daq_memcpy_info_inst.copy_size);
         }
         else
         {
            //memcpy((host->data_buf[b_num]) + (jump_data_num_per_mode + mipi_ch_2_rfic_num[ii] * jump_data_num_per_chip)/2, buffers[2*ii+1][buf.index].start, buf.bytesused);
            daq_memcpy_info_inst.dst = (void*)((host->data_buf[b_num]) + (jump_data_num_per_mode + mipi_ch_2_rfic_num[ii] * jump_data_num_per_chip)/2);
            daq_memcpy_info_inst.src = buffers[2*ii+1][buf.index].start;
            daq_memcpy_info_inst.copy_size = buf.bytesused;
            daq_memcpy_socket.send(&daq_memcpy_info_inst, sizeof(daq_memcpy_info_inst));
            //printf("[SEND] dst: 0x%x, src: 0x%x, size: %d\n", (uint32_t*)daq_memcpy_info_inst.dst, (uint32_t*)daq_memcpy_info_inst.src, daq_memcpy_info_inst.copy_size);
         }*/
           
         // process_image(ii, buffers[buf.index].start, buf.bytesused);
         //printf("ch:%d, ret: %d, size: %d, buf_idx: %d, jump_addr: %d, buff_addr: 0x%X\n",ii,ret,buf.bytesused,buf.index, jump_data_num_per_mode + mipi_ch_2_rfic_num[ii] * jump_data_num_per_chip,buffers[ii][buf.index].start);
         if(host->curr_profile ==0)
            ret = xioctl(host->host_mipi_rfic_dev[2*ii], VIDIOC_QBUF, &buf);
         else
            ret = xioctl(host->host_mipi_rfic_dev[2*ii+1], VIDIOC_QBUF, &buf);
         if (-1 == ret)
            errno_exit("VIDIOC_QBUF");
      }
      break;
   case IO_METHOD_USERPTR:
      CLEAR(buf);

      buf.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
      buf.memory = V4L2_MEMORY_USERPTR;
      for(int ii = 0; ii < 4; ii++)
      {  
         if(host->curr_profile ==0)
            ret = xioctl(host->host_mipi_rfic_dev[2*ii], VIDIOC_DQBUF, &buf);
         else
            ret = xioctl(host->host_mipi_rfic_dev[2*ii+1], VIDIOC_DQBUF, &buf);
         if (-1 == ret)
         {
            switch (errno)
            {
            case EAGAIN:
               return 0;

            case EIO:
               /* Could ignore EIO, see spec. */

               /* fall through */

            default:
               errno_exit("VIDIOC_DQBUF");
            }
         }
         //printf("ch:%d, ret : %d, size : %d, buf_idx : %d, buff_addr : 0x%X\n",ii,ret,buf.bytesused,buf.index, buf.m.userptr);
         if(host->curr_profile ==0)
            ret = xioctl(host->host_mipi_rfic_dev[2*ii], VIDIOC_QBUF, &buf);
         else
            ret = xioctl(host->host_mipi_rfic_dev[2*ii+1], VIDIOC_QBUF, &buf);
         if (-1 == ret)
               errno_exit("VIDIOC_QBUF");
      }
      break;
   }

   return 1;
}


int host_mipi_read_frame(host_device_handle_t *host)
{
   fd_set fdx[HOST_NUM_MIPI_CH];
   struct timeval tv;
   int rx[HOST_NUM_MIPI_CH];
   int isdone = 0;
   int idx = 0;
   int chk = 0;
   int ii = 0;
   int flag;

   for(ii =0; ii<HOST_NUM_MIPI_CH; ii++)
   {  
      FD_ZERO(&fdx[ii]);  
   }
   
   /* Timeout. */
   tv.tv_sec = 5;
   tv.tv_usec = 0;
   flag = 0;
   // printf("host->curr_profile in host_mipi_read_frame : %d \n",host->curr_profile);

   while(flag == 0)
   {
      if(host->curr_profile == 0)
      {
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID0], &fdx[HOST_MIPI_CH0_VCID0]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID0], &fdx[HOST_MIPI_CH1_VCID0]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID0], &fdx[HOST_MIPI_CH2_VCID0]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID0], &fdx[HOST_MIPI_CH3_VCID0]);
         host_rfic_hw_trigger(host);
         // usleep(10);
         rx[HOST_MIPI_CH0_VCID0] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID0]+1, &fdx[HOST_MIPI_CH0_VCID0], NULL, NULL, &tv);
         rx[HOST_MIPI_CH1_VCID0] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID0]+1, &fdx[HOST_MIPI_CH1_VCID0], NULL, NULL, &tv);
         rx[HOST_MIPI_CH2_VCID0] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID0]+1, &fdx[HOST_MIPI_CH2_VCID0], NULL, NULL, &tv);
         rx[HOST_MIPI_CH3_VCID0] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID0]+1, &fdx[HOST_MIPI_CH3_VCID0], NULL, NULL, &tv);
         if ((-1 == rx[HOST_MIPI_CH0_VCID0]) || (-1 == rx[HOST_MIPI_CH1_VCID0]) || (-1 == rx[HOST_MIPI_CH2_VCID0]) || (-1 == rx[HOST_MIPI_CH3_VCID0]) )
         {
            if (EINTR == errno)
            errno_exit("select");
         }

         if ((0 == rx[HOST_MIPI_CH0_VCID0]) || (0 == rx[HOST_MIPI_CH1_VCID0]) || (0 == rx[HOST_MIPI_CH2_VCID0]) || (0 == rx[HOST_MIPI_CH3_VCID0]) )
         {
            fprintf(stderr, "select timeout\n");
            exit(EXIT_FAILURE);
         }
         flag = FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID0], &fdx[HOST_MIPI_CH0_VCID0]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID0], &fdx[HOST_MIPI_CH1_VCID0]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID0], &fdx[HOST_MIPI_CH2_VCID0]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID0], &fdx[HOST_MIPI_CH3_VCID0]);
      }
      else
      {
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID1], &fdx[HOST_MIPI_CH0_VCID1]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID1], &fdx[HOST_MIPI_CH1_VCID1]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID1], &fdx[HOST_MIPI_CH2_VCID1]);
         FD_SET(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID1], &fdx[HOST_MIPI_CH3_VCID1]);
         host_rfic_hw_trigger(host);
         // usleep(10);
         rx[HOST_MIPI_CH0_VCID1] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID1]+1, &fdx[HOST_MIPI_CH0_VCID1], NULL, NULL, &tv);
         rx[HOST_MIPI_CH1_VCID1] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID1]+1, &fdx[HOST_MIPI_CH1_VCID1], NULL, NULL, &tv);
         rx[HOST_MIPI_CH2_VCID1] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID1]+1, &fdx[HOST_MIPI_CH2_VCID1], NULL, NULL, &tv);
         rx[HOST_MIPI_CH3_VCID1] = select(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID1]+1, &fdx[HOST_MIPI_CH3_VCID1], NULL, NULL, &tv);
         if ((-1 == rx[HOST_MIPI_CH0_VCID1]) || (-1 == rx[HOST_MIPI_CH1_VCID1]) || (-1 == rx[HOST_MIPI_CH2_VCID1]) || (-1 == rx[HOST_MIPI_CH3_VCID1]) )
         {
            if (EINTR == errno)
            errno_exit("select");
         }

         if ((0 == rx[HOST_MIPI_CH0_VCID1]) || (0 == rx[HOST_MIPI_CH1_VCID1]) || (0 == rx[HOST_MIPI_CH2_VCID1]) || (0 == rx[HOST_MIPI_CH3_VCID1]) )
         {
            fprintf(stderr, "select timeout\n");
            exit(EXIT_FAILURE);
         }
         flag = FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH0_VCID1], &fdx[HOST_MIPI_CH0_VCID1]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH1_VCID1], &fdx[HOST_MIPI_CH1_VCID1]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH2_VCID1], &fdx[HOST_MIPI_CH2_VCID1]) &
                FD_ISSET(host->host_mipi_rfic_dev[HOST_MIPI_CH3_VCID1], &fdx[HOST_MIPI_CH3_VCID1]);
      }
   }

   host_read_frame(host);

   /* EAGAIN - continue select loop. */
}

int host_mipi_capt_stop(host_device_handle_t *host)
{
   enum v4l2_buf_type type;
   type = V4L2_BUF_TYPE_VIDEO_CAPTURE;

   for(int jj = HOST_MIPI_CH0_VCID0; jj < HOST_NUM_MIPI_CH; jj++)
   {
      if (-1 == xioctl(host->host_mipi_rfic_dev[jj], VIDIOC_STREAMOFF, &type))
         errno_exit("VIDIOC_STREAMOFF");
   }
}

void host_mipi_close(host_device_handle_t *host)
{
   int ii;
   struct stat st;
   int ret;
   

   for (ii = HOST_MIPI_CH0_VCID0; ii < HOST_NUM_MIPI_CH; ii++)
   {
      ret = close(host->host_mipi_rfic_dev[ii]);
      if (-1 == ret)
         errno_exit("close");
   }
}