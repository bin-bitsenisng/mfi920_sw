/*************************************************

File name: cuda_fft.cu
Author: David
Description: CUDA based Range & Doppler FFT including windowing
주의사항: raw data의 배열이 다음과 같다고 가정함. rawdata[SP_NUM_TX][SP_NUM_RX][SP_NUM_CHIRPS][SP_NUM_SAMPLES]

*************************************************/

#include <iostream>
#include <chrono>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <cublasXt.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <unistd.h>
//#include<io.h>
#include <assert.h>
using namespace std;


#define SP_PI				3.141592653589793f

#define SP_NUM_CHIRPS		32//64
#define SP_NUM_TX			12//2
#define SP_NUM_RX			16//8
#define SP_NUM_SAMPLES		1024//1024

#define SP_NUM_FFT_R        SP_NUM_SAMPLES
#define SP_NUM_FFT_R_HALF   (SP_NUM_FFT_R / 2)
#define SP_NUM_FFT_D        SP_NUM_CHIRPS

#define BATCH_SIZE_F_1D		SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX
#define BATCH_SIZE_S_1D		SP_NUM_FFT_R_HALF * SP_NUM_TX * SP_NUM_RX

uint16_t* s16adcData;
hipfftComplex* windowed_1d_data;
hipfftComplex* cf32fftData_1D;
hipfftComplex* windowed_2d_data;
hipfftComplex* transposed_data;
hipfftComplex* cf32fftData_2D;

hipfftComplex* cf32WinCoeff1D;
float* f32WinCoeff2D;

hipfftHandle plan_first_1d_fft;
hipfftHandle plan_second_1d_fft;
hipblasHandle_t cublas_handle;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}

/*********************/
/* CUFFT ERROR CHECK */
/*********************/
static const char *_cudaGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "CUFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "CUFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "CUFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "CUFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "CUFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "CUFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "CUFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "CUFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "CUFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "CUFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
        fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                           _cudaGetErrorEnum(err)); \
        hipDeviceReset(); assert(0); \
    }
}

/**********************/
/* CUBLAS ERROR CHECK */
/**********************/
static const char *_cudaCublasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "CUBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "CUBLAS_STATUS_LICENSE_ERROR";
    }

    return "<unknown>";
}

#define cublasSafeCall(err)      __cublasSafeCall(err, __FILE__, __LINE__)
inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line)
{
    if( HIPBLAS_STATUS_SUCCESS != err) {
        fprintf(stderr, "Cublas error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                           _cudaCublasGetErrorEnum(err)); \
        hipDeviceReset(); assert(0); \
    }
}

__global__ void window_1d_coeff_mul(uint16_t* src, hipfftComplex* dst, hipfftComplex* coeff)
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int offset = bi * blockDim.x;

	dst[offset + ti].x = (float)(src[offset + ti]) * coeff[ti].x;
	dst[offset + ti].y = 0.0f;
}

__global__ void window_2d_coeff_mul(hipfftComplex* src, hipfftComplex* dst, int size, float* coeff)
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int dst_offset = bi*size;
	int src_offset = dst_offset*2;

	dst[dst_offset + ti].x = src[src_offset + ti].x * coeff[bi];
	dst[dst_offset + ti].y = src[src_offset + ti].y * coeff[bi];
}

int FFT_2d_cuda(void)
{	
	chrono::system_clock::time_point time_start, time_end, total_time_start, total_time_end;

	// Init temporal input data	
	for (int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for (int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_CHIRPS; di++)
			{
				for (int ri = 0; ri < SP_NUM_SAMPLES; ri++)
				{
					s16adcData[SP_NUM_CHIRPS*SP_NUM_SAMPLES*(tx*SP_NUM_RX+rx) + SP_NUM_SAMPLES*di + ri] = (ri + rx + tx + di) * 10;
				}
			}
		}
	}

	total_time_start = chrono::system_clock::now();
	
	// time_start = chrono::system_clock::now();
	// First Windowing
	window_1d_coeff_mul<<<SP_NUM_CHIRPS*SP_NUM_TX*SP_NUM_RX, SP_NUM_SAMPLES>>>(s16adcData, windowed_1d_data, cf32WinCoeff1D);

	// cudaStreamAttachMemAsync(NULL, windowed_1d_data, 0, cudaMemAttachHost);
	// cudaStreamSynchronize(NULL);	
	// time_end = chrono::system_clock::now();
	// cout << "First Windowing takes " << chrono::duration_cast<chrono::microseconds>(time_end - time_start).count() << " us" << endl << endl;

#ifdef DATA_LOG
	cudaStreamAttachMemAsync(NULL, windowed_1d_data, 0, cudaMemAttachHost);
	cudaStreamSynchronize(NULL);

	printf("windowed 1d data..\n");

	for(int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for(int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_CHIRPS; di++)
			{
				for (int ri = 0; ri < SP_NUM_SAMPLES; ri++)
				{
					printf("[%f, %f]\n", windowed_1d_data[SP_NUM_CHIRPS*SP_NUM_SAMPLES*(tx*SP_NUM_RX+rx) + SP_NUM_SAMPLES*di + ri].x, \
										windowed_1d_data[SP_NUM_CHIRPS*SP_NUM_SAMPLES*(tx*SP_NUM_RX+rx) + SP_NUM_SAMPLES*di + ri].y);
				}
				printf("\n");
			}
		}
	}
#endif

	// time_start = chrono::system_clock::now();
	// First 1D fft
	cufftSafeCall(hipfftExecC2C(plan_first_1d_fft, windowed_1d_data, cf32fftData_1D, HIPFFT_FORWARD));
	// cudaStreamAttachMemAsync(NULL, cf32fftData_1D, 0, cudaMemAttachHost);
	// cudaStreamSynchronize(NULL);
	// time_end = chrono::system_clock::now();
	// cout << "First FFT takes " << chrono::duration_cast<chrono::microseconds>(time_end - time_start).count() << " us" << endl << endl;
	
#ifdef DATA_LOG
	// Prefetch 'fft_1d_result' to CPU as just to check data
	cudaStreamAttachMemAsync(NULL, cf32fftData_1D, 0, cudaMemAttachHost);
	cudaStreamSynchronize(NULL);

	printf("fft_1d_result..\n");

	for(int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for(int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_CHIRPS; di++)
			{
				for (int ri = 0; ri < SP_NUM_SAMPLES; ri++)
				{
					printf("[%f, %f]\n", cf32fftData_1D[SP_NUM_CHIRPS*SP_NUM_SAMPLES*(tx*SP_NUM_RX+rx) + SP_NUM_SAMPLES*di + ri].x, \
									cf32fftData_1D[SP_NUM_CHIRPS*SP_NUM_SAMPLES*(tx*SP_NUM_RX+rx) + SP_NUM_SAMPLES*di + ri].y);
				}
				printf("\n");
			}
		}
	}
#endif

	// time_start = chrono::system_clock::now();
	// Second Windowing
	window_2d_coeff_mul<<<SP_NUM_FFT_D*SP_NUM_TX*SP_NUM_RX, SP_NUM_FFT_R_HALF>>>(cf32fftData_1D, windowed_2d_data, SP_NUM_FFT_R_HALF, f32WinCoeff2D);

	// cudaStreamAttachMemAsync(NULL, windowed_2d_data, 0, cudaMemAttachHost);
	// cudaStreamSynchronize(NULL);
	// time_end = chrono::system_clock::now();
	// cout << "Second Windowing takes " << chrono::duration_cast<chrono::microseconds>(time_end - time_start).count() << " us" << endl << endl;

#ifdef DATA_LOG
	// Prefetch 'windowed_2d_data' to CPU as just to check data
	cudaStreamAttachMemAsync(NULL, windowed_2d_data, 0, cudaMemAttachHost);
	cudaStreamSynchronize(NULL);

	printf("windowed_2d_data..\n");
	for(int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for(int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_FFT_D; di++)
			{
				for (int ri = 0; ri < SP_NUM_FFT_R_HALF; ri++)
				{
					printf("[%f, %f]\n", windowed_2d_data[SP_NUM_FFT_D*SP_NUM_FFT_R_HALF*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_R_HALF + ri].x, \
											windowed_2d_data[SP_NUM_FFT_D*SP_NUM_FFT_R_HALF*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_R_HALF + ri].y);
				}
				printf("\n");
			}
		}
	}
#endif

	// Transpose
	hipComplex alpha;
	hipComplex beta;
	alpha.x = 1.0; alpha.y = 0.0;
	beta.x = 0.0; beta.y = 0.0;

	//time_start = chrono::system_clock::now();
	for(int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for(int rx = 0; rx < SP_NUM_RX; rx++)
		{
			cublasSafeCall(hipblasCgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, SP_NUM_CHIRPS, SP_NUM_FFT_R_HALF, &alpha, windowed_2d_data + SP_NUM_FFT_D*SP_NUM_FFT_R_HALF*(tx*SP_NUM_RX+rx), SP_NUM_FFT_R_HALF, &beta, NULL, SP_NUM_CHIRPS, transposed_data + SP_NUM_FFT_D*SP_NUM_FFT_R_HALF*(tx*SP_NUM_RX+rx), SP_NUM_CHIRPS));
		}
	}
	// cudaStreamAttachMemAsync(NULL, transposed_data, 0, cudaMemAttachHost);
	// cudaStreamSynchronize(NULL);
	// time_end = chrono::system_clock::now();
	// cout << "Transpose takes " << chrono::duration_cast<chrono::microseconds>(time_end - time_start).count() << " us" << endl << endl;

#ifdef DATA_LOG
	// Prefetch 'transposed_data' to CPU as just to check data	
	cudaStreamAttachMemAsync(NULL, transposed_data, 0, cudaMemAttachHost);
	cudaStreamSynchronize(NULL);

	printf("transposed_data..\n");
	for(int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for(int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_FFT_R_HALF; di++)
			{
				for (int ri = 0; ri < SP_NUM_FFT_D; ri++)
				{
					printf("[%f, %f]\n", transposed_data[SP_NUM_FFT_R_HALF*SP_NUM_FFT_D*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_D + ri].x, \
											transposed_data[SP_NUM_FFT_R_HALF*SP_NUM_FFT_D*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_D + ri].y);
				}
				printf("\n");
			}
		}
	}
#endif

	// time_start = chrono::system_clock::now();
	// Second 1D fft 
	cufftSafeCall(hipfftExecC2C(plan_second_1d_fft, transposed_data, cf32fftData_2D, HIPFFT_FORWARD));

	gpuErrchk(hipStreamAttachMemAsync(NULL, cf32fftData_2D, 0, hipMemAttachHost));
	gpuErrchk(hipStreamSynchronize(NULL));

	//time_end = chrono::system_clock::now();
	//cout << "Second FFT takes " << chrono::duration_cast<chrono::microseconds>(time_end - time_start).count() << " us" << endl << endl;
	total_time_end = chrono::system_clock::now();	
	cout << "2D FFT All takes " << chrono::duration_cast<chrono::microseconds>(total_time_end - total_time_start).count() << " us" << endl;

#ifdef DATA_LOG
	printf("2D FFT result..\n");
	for (int tx = 0; tx < SP_NUM_TX; tx++)
	{
		for (int rx = 0; rx < SP_NUM_RX; rx++)
		{
			for (int di = 0; di < SP_NUM_FFT_R_HALF; di++)
			{
				for (int ri = 0; ri < SP_NUM_FFT_D; ri++)
				{
					printf("[%f, %f]\n", cf32fftData_2D[SP_NUM_FFT_R_HALF*SP_NUM_FFT_D*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_D + ri].x, \
										cf32fftData_2D[SP_NUM_FFT_R_HALF*SP_NUM_FFT_D*(tx*SP_NUM_RX+rx) + di*SP_NUM_FFT_D + ri].y);
				}
				printf("\n");
			}
		}
	}
#endif

    return 0;
}

void Init_cuda(void)
{
	size_t sizeOfs16adcData = SP_NUM_CHIRPS*SP_NUM_TX*SP_NUM_RX*SP_NUM_SAMPLES*sizeof(int16_t); // Radar raw data
	size_t sizeOfWindowed1dData = SP_NUM_CHIRPS*SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_R*sizeof(hipfftComplex); // Windowed raw data	
	size_t sizeOfcf32fftData_1D = SP_NUM_CHIRPS*SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_R*sizeof(hipfftComplex); // Result of first(range) FFT data
	size_t sizeOfWindowed2dData = SP_NUM_FFT_R_HALF*SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_D*sizeof(hipfftComplex); // Windowed range FFT data
	size_t sizeOfTransposedData = SP_NUM_FFT_R_HALF*SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_D*sizeof(hipfftComplex); // Transposed windowed range FFT data
	size_t sizeOfcf32fftData_2D = SP_NUM_FFT_R_HALF*SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_D*sizeof(hipfftComplex); // Final result of second(doppler) FFT data
	gpuErrchk(hipMallocManaged((void**)&s16adcData, sizeOfs16adcData));
	gpuErrchk(hipMallocManaged((void**)&windowed_1d_data, sizeOfWindowed1dData));
	gpuErrchk(hipMallocManaged((void**)&cf32fftData_1D, sizeOfcf32fftData_1D));
	gpuErrchk(hipMallocManaged((void**)&windowed_2d_data, sizeOfWindowed2dData));
	gpuErrchk(hipMallocManaged((void**)&transposed_data, sizeOfTransposedData));
	gpuErrchk(hipMallocManaged((void**)&cf32fftData_2D, sizeOfcf32fftData_2D));

    size_t sizeOfcf32WinCoeff1D = SP_NUM_FFT_R*sizeof(hipfftComplex); // Hamming window coefficient for raw data
    size_t sizeOff32WinCoeff2D = SP_NUM_TX*SP_NUM_RX*SP_NUM_FFT_D*sizeof(float); // Hamming window coefficient for range FFT data
    gpuErrchk(hipMallocManaged((void**)&cf32WinCoeff1D, sizeOfcf32WinCoeff1D));
    gpuErrchk(hipMallocManaged((void**)&f32WinCoeff2D, sizeOff32WinCoeff2D));

    cufftSafeCall(hipfftPlan1d(&plan_first_1d_fft, SP_NUM_SAMPLES, HIPFFT_C2C, BATCH_SIZE_F_1D));
	cufftSafeCall(hipfftPlan1d(&plan_second_1d_fft, SP_NUM_FFT_D, HIPFFT_C2C, BATCH_SIZE_S_1D));
	
	cublasSafeCall(hipblasCreate(&cublas_handle));
}

void Gen_wincoeff(void) // SP_CAL_SAMPLE_NUM == 0xF014 case cannot be covered.
{
	int16_t idx;
	float phi;
	float temp_coef;
	float norm_factor;

	norm_factor = 1.0f / 2048.0f;
	phi = 2.0f * SP_PI / (float)(SP_NUM_FFT_R - 1);

	for (idx = 0; idx < SP_NUM_FFT_R; idx++)
	{
		/* Hamming window */
		temp_coef = 0.53836f - (0.46164f * cosf(phi * (float)idx));

		cf32WinCoeff1D[idx].x = temp_coef * norm_factor;
	}

	phi = 2.0f * SP_PI / (float)(SP_NUM_FFT_D - 1);
	for (int txrx = 0; txrx < SP_NUM_TX*SP_NUM_RX; txrx++)
	{	
		for (idx = 0; idx < SP_NUM_FFT_D; idx++)
		{
			f32WinCoeff2D[txrx*SP_NUM_FFT_D + idx] = 0.53836f - (0.46164f * cosf(phi * (float)idx));
		}
	}
}

int main(void)
{
	Init_cuda();
	Gen_wincoeff();

	while(1)
	{
		FFT_2d_cuda();
		//usleep(30000);
		usleep(50000);
	}
}
