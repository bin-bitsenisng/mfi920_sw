#include "hip/hip_runtime.h"
/**************************************************************************************
* -----------------------------------   include   ----------------------------------- *
**************************************************************************************/
#include "Obj_Main.h"
#include "Pre_Main.h"
#include "Obj_AngEst.h"
#include "Obj_Parameters.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include ""
#ifdef LINUX
#include <sys/time.h>
#include "print_common.h"
#endif
/**************************************************************************************
* -----------------------------------   define   ------------------------------------ *
**************************************************************************************/
#define MAX_NUM_OF_GPU_THREAD	1024

/**************************************************************************************
* ------------------------------   global variables   ------------------------------- *
**************************************************************************************/
#ifdef CUDA_ACC
hipComplex* ch_fft_lr;
hipComplex* ch_fft_mr;
hipComplex* ch_fft_sr;

hipComplex* cmplx_element;

hipComplex* cmplx_sum_lr;
hipComplex* cmplx_sum_mr;
hipComplex* cmplx_sum_sr;

float32_t* temp_aps_lr;
float32_t* temp_aps_mr;
float32_t* temp_aps_sr;


//float32_t* temp_aps2;
#endif

hipblasHandle_t cublas_handler;

/**************************************************************************************
* -----------------------------   function prototypes   ----------------------------- *
**************************************************************************************/
#ifdef CUDA_ACC
static void Obj_vAngleEstimation_LR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod);
static void Obj_vAngleEstimation_MR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod);
static void Obj_vAngleEstimation_SR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod);
#else
//static void Obj_vAngleEstimation_LR(PeakData_t *PeakData, MultiMod_t CurMod);
//static void Obj_vAngleEstimation_MR(PeakData_t *PeakData, MultiMod_t CurMod);
//static void Obj_vAngleEstimation_SR(PeakData_t *PeakData, MultiMod_t CurMod);
#endif
//static void Obj_vAngleEstimation_Ref(RefPeakData_t * PeakData);

static cmplxf32_t Obj_cf32GetPhaseDiff(const cmplxf32_t a, const cmplxf32_t b);
//static cmplxf32_t Obj_cf32DivPhase(const cmplxf32_t a, const cmplxf32_t b);
static void Obj_vAryRemov(cmplxf32_t temp_peak_fft[], cmplxf32_t temp_ch_fft[]);
static void Obj_vAryPaste(cmplxf32_t temp_peak_fft[], cmplxf32_t temp_ch_fft[]);

/**************************************************************************************
* ------------------------------   global functions   ------------------------------- *
**************************************************************************************/

__global__ void cal_temp_aps(float32_t* temp_aps, hipComplex* cmplx_sum, float32_t ch_norm_factor)
{
	int ti = threadIdx.x;
	int bd = blockDim.x * blockIdx.x;

	temp_aps[bd + ti] = (cmplx_sum[bd + ti].x * cmplx_sum[bd + ti].x + cmplx_sum[bd + ti].y * cmplx_sum[bd + ti].y) * ch_norm_factor;
}

void Obj_vAngEst(void)
{
#ifdef LINUX
	struct timeval startTime, endTime;
	double diffTime;
#endif   

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

#ifdef LINUX
	gettimeofday(&endTime, NULL);
	diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
	//TRACE_TIME_MSG("Obj_vAngleEstimation_Ref(&gRM_PeakData): %lf s\n", diffTime);
#endif	

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	Obj_vAngleEstimation_LR(&gLR_PeakData, LR_MOD);

#ifdef LINUX
	gettimeofday(&endTime, NULL);
	diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
	//TRACE_TIME_MSG("Obj_vAngleEstimation(&gLR_PeakData, LR_MOD)(): %lf s\n", diffTime);
#endif

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	Obj_vAngleEstimation_MR(&gMR_PeakData, MR_MOD);

#ifdef LINUX
	gettimeofday(&endTime, NULL);
	diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
	//TRACE_TIME_MSG("Obj_vAngleEstimation(&gMR_PeakData, MR_MOD)(): %lf s\n", diffTime);
#endif	

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	Obj_vAngleEstimation_SR(&gSR_PeakData, SR_MOD);

#ifdef LINUX
	gettimeofday(&endTime, NULL);
	diffTime = endTime.tv_sec + endTime.tv_usec / 1000000.0 - startTime.tv_sec - startTime.tv_usec / 1000000.0;
	//TRACE_TIME_MSG("Obj_vAngleEstimation(&gSR_PeakData, SR_MOD)(): %lf s\n", diffTime);
#endif
}


/**************************************************************************************
* ------------------------------   static functions   ------------------------------- *
**************************************************************************************/
#ifdef CUDA_ACC
static void Obj_vAngleEstimation_LR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod)
#else
static void Obj_vAngleEstimation(PeakData_t *PeakData, MultiMod_t CurMod)
#endif 
{
#ifdef LINUX
	struct timeval startTime, endTime;
	double diffTime;
#endif  

	uint16_t i, azi_idx, ele_idx, rx, tx, ch, j, ii, k;


	float32_t temp_abs_sum;
	float32_t aps_sll_th[OBJ_NUM_PEAK_HARM_MAX][OBJ_CO_NUM_HARMONICS_MAX];

	float32_t num_azi, den_azi, dazi;
	float32_t num_ele, den_ele, dele;
	float32_t ch_norm_factor;

	float32_t Peak_Candi[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_azi_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_ele_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_near_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];

	int16_t peak_cnt;
	int16_t ang_cnt;

	float32_t temp_peak;
	int16_t temp_azi_idx;
	int16_t temp_ele_idx;
	int16_t temp_peak_flg;

	int16_t temp_peak_azi_idx, temp_peak_ele_idx;

	cmplxf32_t temp_fft[OBJ_AE_NUM_CH_PARALLE];

	ch_norm_factor = _sp_pow2(1.0f / OBJ_AE_NUM_CH);

	int16_t azi_num_aps_step;
	int16_t ele_num_aps_step;
	float32_t sll_th;
	float32_t aps_multi_th;
	float32_t azi_aps_step_size;
	float32_t ele_aps_step_size;
	float32_t chirp_interval;
	float32_t tdm_comp_constant;
	uint32_t temp_index;
	uint32_t pre_temp_index;
	int16_t  num_azi_ele;


	float32_t azi_offset, ele_offset;
	float32_t azi_diff, ele_diff;

	int16_t temp_peak_cnt;
	float32_t peak_azi_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];
	float32_t peak_ele_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];


	azi_num_aps_step = OBJ_AE_AZI_LR_NUM_APS_STEPS;
	ele_num_aps_step = OBJ_AE_ELE_LR_NUM_APS_STEPS;
	sll_th = OBJ_AE_LR_APS_SLL_TH;
	aps_multi_th = OBJ_AE_LR_MULTI_TARGET_TH;
	azi_aps_step_size = OBJ_AE_AZI_LR_APS_STEP_SIZE;
	ele_aps_step_size = OBJ_AE_ELE_LR_APS_STEP_SIZE;
	chirp_interval = SP_LR_CHIRP_INTERVAL;

	num_azi_ele = azi_num_aps_step * ele_num_aps_step;

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
#if (OBJ_HEXA_ARRAY_ON==1)

#ifdef CUDA_ACC
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], (cmplxf32_t*)ch_fft_lr + i * OBJ_AE_NUM_CH);
#else
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], ch_fft[i][k]);
#endif

#else

#endif
			if (PeakData->rdu16phFreqIdx[i].D == 0)
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i]*0.5f);
			}
			else
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i]);
			}

			aps_sll_th[i][k] = (temp_abs_sum* (float32_t)OBJ_AE_NUM_CH) * sll_th * ch_norm_factor;
		}
	}

	hipComplex alpha;
	hipComplex beta;
	alpha.x = 1.0; alpha.y = 0.0;
	beta.x = 0.0; beta.y = 0.0;

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
#ifndef CUDA_ACC
			for (ele_idx = 0; ele_idx < ele_num_aps_step; ele_idx++)
			{
				for (azi_idx = 0; azi_idx < azi_num_aps_step; azi_idx++)
				{
					cmplx_sum.real = 0.0f;
					cmplx_sum.imag = 0.0f;
					for (ch = 0; ch < OBJ_AE_NUM_CH; ch++)
					{
						cmplx_sum.real += (ch_fft[i][k][ch].real * PeakData->cf32SteerVec[ele_idx * azi_num_aps_step + azi_idx][ch].real) + (ch_fft[i][k][ch].imag * PeakData->cf32SteerVec[ele_idx * azi_num_aps_step + azi_idx][ch].imag);
						cmplx_sum.imag += (ch_fft[i][k][ch].imag * PeakData->cf32SteerVec[ele_idx * azi_num_aps_step + azi_idx][ch].real) - (ch_fft[i][k][ch].real * PeakData->cf32SteerVec[ele_idx * azi_num_aps_step + azi_idx][ch].imag);
					}
					temp_aps[i][k][ele_idx][azi_idx] = ((cmplx_sum.real * cmplx_sum.real) + (cmplx_sum.imag * cmplx_sum.imag)) * ch_norm_factor;
				}
			}
#else
			hipblasCgemv(cublas_handler, HIPBLAS_OP_T, OBJ_AE_NUM_CH, num_azi_ele, &alpha, (hipComplex*)PeakData->cf32SteerVec, OBJ_AE_NUM_CH, \
				ch_fft_lr + OBJ_AE_NUM_CH * (i), 1, &beta, cmplx_sum_lr + num_azi_ele * (i), 1);
#endif
		}
	}


#ifdef CUDA_ACC
	cal_temp_aps << < (uint32_t)(PeakData->s16PhCnt * num_azi_ele / MAX_NUM_OF_GPU_THREAD) + 1, \
		MAX_NUM_OF_GPU_THREAD >> > (temp_aps_lr, cmplx_sum_lr, ch_norm_factor);
#ifdef LINUX
	(hipStreamAttachMemAsync(NULL, temp_aps_lr, 0, hipMemAttachHost));
#else
	(hipStreamAttachMemAsync(NULL, temp_aps_lr, 0, hipMemAttachGlobal));
#endif
	(hipStreamSynchronize(NULL));
#endif

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
			peak_cnt = 0;
			temp_peak_azi_idx = 0;
			temp_peak_ele_idx = 0;
			
			for (ele_idx = 1; ele_idx < ele_num_aps_step - 1; ele_idx++)
			{
				_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);
				for (azi_idx = 1; azi_idx < azi_num_aps_step - 1; azi_idx++)
				{
					_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);
					temp_index = i * num_azi_ele + ele_idx * azi_num_aps_step;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
					if ((temp_aps_lr[temp_index + azi_idx] > aps_sll_th[i][k]))
#else
					if ((temp_aps_lr[temp_index + azi_idx] > aps_sll_th[i][k])
						&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_idx - 1])
						&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_idx + 1])
						&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index - azi_num_aps_step + azi_idx])
						&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_num_aps_step + azi_idx]))
#endif
					{
						_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);

						Peak_Candi[peak_cnt] = temp_aps_lr[temp_index + azi_idx];
						Peak_azi_idx[peak_cnt] = azi_idx;
						Peak_ele_idx[peak_cnt] = ele_idx;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
						if ((temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_idx - 1])
							&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_idx + 1])
							&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index - azi_num_aps_step + azi_idx])
							&& (temp_aps_lr[temp_index + azi_idx] > temp_aps_lr[temp_index + azi_num_aps_step + azi_idx]))
						{
							Peak_flg[peak_cnt] = 1U;
						}
						else
						{
							Peak_flg[peak_cnt] = 0U;
						}
#else
						Peak_flg[peak_cnt] = 1U;
#endif
						peak_cnt++;
					}
				}
			}

			for (ii = 0; ii < peak_cnt; ++ii)
			{
				for (j = ii + 1; j < peak_cnt; ++j)
				{

					if (Peak_Candi[ii] < Peak_Candi[j])
					{
						temp_peak = Peak_Candi[ii];
						Peak_Candi[ii] = Peak_Candi[j];
						Peak_Candi[j] = temp_peak;

						temp_azi_idx = Peak_azi_idx[ii];
						Peak_azi_idx[ii] = Peak_azi_idx[j];
						Peak_azi_idx[j] = temp_azi_idx;

						temp_ele_idx = Peak_ele_idx[ii];
						Peak_ele_idx[ii] = Peak_ele_idx[j];
						Peak_ele_idx[j] = temp_ele_idx;

						temp_peak_flg = Peak_flg[ii];
						Peak_flg[ii] = Peak_flg[j];
						Peak_flg[j] = temp_peak_flg;
					}
				}
			}

			temp_peak_cnt = 0;
			for (j = 0; j < peak_cnt; j++)
			{
				if (Peak_flg[j] == 1)
				{
					_sp_cnt_break(temp_peak_cnt, OBJ_AE_NUM_PEAK_TARGET_MAX);
					peak_azi_temp_idx[temp_peak_cnt] = Peak_azi_idx[j];
					peak_ele_temp_idx[temp_peak_cnt] = Peak_ele_idx[j];
					temp_peak_cnt++;
				}
			}

			for (j = 0; j < peak_cnt; j++)
			{
				for (ii = 0; ii < temp_peak_cnt; ii++)
				{
					azi_diff = _sp_pow2(_sp_abs(peak_azi_temp_idx[ii] - Peak_azi_idx[j]));
					ele_diff = _sp_pow2(_sp_abs(peak_ele_temp_idx[ii] - Peak_ele_idx[j]));

					if (_sp_sqrtf(azi_diff + ele_diff) < 4.0f)
					{
						Peak_near_flg[j] = 1;
					}
				}
			}

			ang_cnt = 0;
			PeakData->s16AngCnt[i][k] = 0;
			if (peak_cnt > 0)
			{
				for (j = 0; j < peak_cnt; j++)
				{
					if ((Peak_Candi[j] > aps_multi_th*Peak_Candi[0])
						&& (Peak_near_flg[j] == 1))
					{

						_sp_cnt_break(ang_cnt, OBJ_AE_NUM_TARGET_MAX);
						temp_index = i * num_azi_ele;

						num_azi = temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];
						den_azi = temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - (2.0f * temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];

						num_ele = temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];
						den_ele = temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - (2.0f * temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];

						dazi = 0.5f * azi_aps_step_size;
						dele = 0.5f * ele_aps_step_size;

						azi_offset = (num_azi / den_azi) * dazi;
						ele_offset = (num_ele / den_ele) * dele;
						
						if (_sp_abs(azi_offset) < 1.0f)
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]] + azi_offset;
						}
						else
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]];
						}

						if (_sp_abs(ele_offset) < 2.0f)
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]] + ele_offset;
						}
						else
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]];
						}

						PeakData->f32Power[i][k][ang_cnt] = 10.0f*_sp_log10f(temp_aps_lr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]);

						if (Peak_flg[j] == 1U)
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 1;
						}
						else
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 0;
						}
						ang_cnt++;
					}
				}
			}
			else
			{
				PeakData->f32AziAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32EleAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32Power[i][k][0] = 0.0f;
			}

			PeakData->s16AngCnt[i][k] = ang_cnt;

			/* for Debug */
#ifdef WIN32
			for (ele_idx = 0; ele_idx < ele_num_aps_step; ele_idx++)
			{
				for (azi_idx = 0; azi_idx < azi_num_aps_step; azi_idx++)
				{
					PeakData->f32APS[i][ele_idx * azi_num_aps_step + azi_idx] = temp_aps_lr[i * num_azi_ele + ele_idx * azi_num_aps_step + azi_idx];
				}
			}
#endif
		}
	}
}

#ifdef CUDA_ACC
static void Obj_vAngleEstimation_MR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod)
#else
static void Obj_vAngleEstimation_MR(PeakData_t *PeakData, MultiMod_t CurMod)
#endif 
{
#ifdef LINUX
	struct timeval startTime, endTime;
	double diffTime;
#endif  

	uint16_t i, azi_idx, ele_idx, rx, tx, ch, j, ii, k;


	float32_t temp_abs_sum;
	float32_t aps_sll_th[OBJ_NUM_PEAK_HARM_MAX][OBJ_CO_NUM_HARMONICS_MAX];

	float32_t num_azi, den_azi, dazi;
	float32_t num_ele, den_ele, dele;
	float32_t ch_norm_factor;

	float32_t Peak_Candi[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_azi_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_ele_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_near_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];

	int16_t peak_cnt;
	int16_t ang_cnt;

	float32_t temp_peak;
	int16_t temp_azi_idx;
	int16_t temp_ele_idx;
	int16_t temp_peak_azi_idx;
	int16_t temp_peak_ele_idx;
	int16_t temp_peak_flg;

	cmplxf32_t temp_fft[OBJ_AE_NUM_CH_PARALLE];

	ch_norm_factor = _sp_pow2(1.0f / OBJ_AE_NUM_CH);

	int16_t azi_num_aps_step;
	int16_t ele_num_aps_step;
	float32_t sll_th;
	float32_t aps_multi_th;
	float32_t azi_aps_step_size;
	float32_t ele_aps_step_size;
	float32_t chirp_interval;
	float32_t tdm_comp_constant;
	uint32_t temp_index;
	uint32_t pre_temp_index;
	int16_t num_azi_ele;

	float32_t azi_offset, ele_offset;
	float32_t azi_diff, ele_diff;
	
	int16_t temp_peak_cnt;
	float32_t peak_azi_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];
	float32_t peak_ele_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];



	azi_num_aps_step = OBJ_AE_AZI_MR_NUM_APS_STEPS;
	ele_num_aps_step = OBJ_AE_ELE_MR_NUM_APS_STEPS;
	sll_th = OBJ_AE_MR_APS_SLL_TH;
	aps_multi_th = OBJ_AE_MR_MULTI_TARGET_TH;
	azi_aps_step_size = OBJ_AE_AZI_MR_APS_STEP_SIZE;
	ele_aps_step_size = OBJ_AE_ELE_MR_APS_STEP_SIZE;
	chirp_interval = SP_MR_CHIRP_INTERVAL;

	num_azi_ele = azi_num_aps_step * ele_num_aps_step;

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{

		for (k = 0; k < 1; k++)
		{

#if (OBJ_HEXA_ARRAY_ON==1)

#ifdef CUDA_ACC
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], (cmplxf32_t*)ch_fft_mr + i * OBJ_AE_NUM_CH);
#else
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], ch_fft[i][k]);
#endif

#else

#ifdef CUDA_ACC
			Obj_vAryPaste(temp_fft[i][k], (cmplxf32_t*)ch_fft + i * OBJ_AE_NUM_CH);
#else
			Obj_vAryPaste(temp_fft[i][k], ch_fft[i][k]);
#endif

#endif
			if (PeakData->rdu16phFreqIdx[i].D == 0)
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i] * 0.5f);
			}
			else
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i]);
			}
			aps_sll_th[i][k] = (temp_abs_sum * (float32_t)OBJ_AE_NUM_CH) * sll_th * ch_norm_factor;
		}
	}

	hipComplex alpha;
	hipComplex beta;
	alpha.x = 1.0; alpha.y = 0.0;
	beta.x = 0.0; beta.y = 0.0;

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
			hipblasCgemv(cublas_handler, HIPBLAS_OP_T, OBJ_AE_NUM_CH, num_azi_ele, &alpha, (hipComplex*)PeakData->cf32SteerVec, OBJ_AE_NUM_CH, \
				ch_fft_mr + OBJ_AE_NUM_CH * (i), 1, &beta, cmplx_sum_mr + num_azi_ele * i, 1);

		}
	}


#ifdef CUDA_ACC
	cal_temp_aps << < (uint32_t)(PeakData->s16PhCnt* num_azi_ele / MAX_NUM_OF_GPU_THREAD) + 1, MAX_NUM_OF_GPU_THREAD >> > (temp_aps_mr, cmplx_sum_mr, ch_norm_factor);
#ifdef LINUX
	(hipStreamAttachMemAsync(NULL, temp_aps_mr, 0, hipMemAttachHost));
#else
	(hipStreamAttachMemAsync(NULL, temp_aps_mr, 0, hipMemAttachGlobal));
#endif
	(hipStreamSynchronize(NULL));
#endif


	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
			peak_cnt = 0;
			temp_peak_azi_idx = 0;
			temp_peak_ele_idx = 0;

			for (ele_idx = 1; ele_idx < ele_num_aps_step - 1; ele_idx++)
			{
				_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);
				for (azi_idx = 1; azi_idx < azi_num_aps_step - 1; azi_idx++)
				{

					_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);

					temp_index = i * num_azi_ele + ele_idx * azi_num_aps_step;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
					if ((temp_aps_mr[temp_index + azi_idx] > aps_sll_th[i][k]))
#else
					if ((temp_aps_mr[temp_index + azi_idx] > aps_sll_th[i][k])
						&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_idx - 1])
						&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_idx + 1])
						&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index - azi_num_aps_step + azi_idx])
						&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_num_aps_step + azi_idx]))
#endif
					{
						_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);

						Peak_Candi[peak_cnt] = temp_aps_mr[temp_index + azi_idx];
						Peak_azi_idx[peak_cnt] = azi_idx;
						Peak_ele_idx[peak_cnt] = ele_idx;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
						if ((temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_idx - 1])
							&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_idx + 1])
							&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index - azi_num_aps_step + azi_idx])
							&& (temp_aps_mr[temp_index + azi_idx] > temp_aps_mr[temp_index + azi_num_aps_step + azi_idx]))
						{
							Peak_flg[peak_cnt] = 1U;
						}
						else
						{
							Peak_flg[peak_cnt] = 0U;
						}
#else
						Peak_flg[peak_cnt] = 1U;
#endif
						peak_cnt++;
					}
				}
			}

			for (ii = 0; ii < peak_cnt; ++ii)
			{
				for (j = ii + 1; j < peak_cnt; ++j)
				{

					if (Peak_Candi[ii] < Peak_Candi[j])
					{
						temp_peak = Peak_Candi[ii];
						Peak_Candi[ii] = Peak_Candi[j];
						Peak_Candi[j] = temp_peak;

						temp_azi_idx = Peak_azi_idx[ii];
						Peak_azi_idx[ii] = Peak_azi_idx[j];
						Peak_azi_idx[j] = temp_azi_idx;

						temp_ele_idx = Peak_ele_idx[ii];
						Peak_ele_idx[ii] = Peak_ele_idx[j];
						Peak_ele_idx[j] = temp_ele_idx;

						temp_peak_flg = Peak_flg[ii];
						Peak_flg[ii] = Peak_flg[j];
						Peak_flg[j] = temp_peak_flg;
					}
				}
			}

			temp_peak_cnt = 0;
			for( j = 0 ; j < peak_cnt; j++)
			{
				if (Peak_flg[j] == 1)
				{
					_sp_cnt_break(temp_peak_cnt, OBJ_AE_NUM_PEAK_TARGET_MAX);
					peak_azi_temp_idx[temp_peak_cnt] = Peak_azi_idx[j];
					peak_ele_temp_idx[temp_peak_cnt] = Peak_ele_idx[j];
					temp_peak_cnt++;
				}
			}

			for (j = 0; j < peak_cnt; j++)
			{
				for (ii = 0; ii < temp_peak_cnt; ii++)
				{
					azi_diff = _sp_pow2(_sp_abs(peak_azi_temp_idx[ii] - Peak_azi_idx[j]));
					ele_diff = _sp_pow2(_sp_abs(peak_ele_temp_idx[ii] - Peak_ele_idx[j]));

					if (_sp_sqrtf(azi_diff + ele_diff) < 4.0f)
					{
						Peak_near_flg[j] = 1;
					}
				}
			}

			ang_cnt = 0;
			PeakData->s16AngCnt[i][k] = 0;
			if (peak_cnt > 0)
			{
				for (j = 0; j < peak_cnt; j++)
				{
					if ((Peak_Candi[j] > aps_multi_th*Peak_Candi[0]) 
						&& (Peak_near_flg[j] == 1))
					{

						_sp_cnt_break(ang_cnt, OBJ_AE_NUM_TARGET_MAX);
						temp_index = i * ele_num_aps_step*azi_num_aps_step;

						num_azi = temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];
						den_azi = temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - (2.0f * temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];

						num_ele = temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];
						den_ele = temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - (2.0f * temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];

						dazi = 0.5f * azi_aps_step_size;
						dele = 0.5f * ele_aps_step_size;

						azi_offset = (num_azi / den_azi) * dazi;
						ele_offset = (num_ele / den_ele) * dele;
						
						if (_sp_abs(azi_offset) < 1.0f)
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]] + azi_offset;
						}
						else
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]];
						}

						if (_sp_abs(ele_offset) < 2.0f)
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]] + ele_offset;
						}
						else
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]];
						}

						PeakData->f32Power[i][k][ang_cnt] = 10.0f*_sp_log10f(temp_aps_mr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]);

						if (Peak_flg[j] == 1U)
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 1;
						}
						else
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 0;
						}

						ang_cnt++;
					}
				}
			}
			else
			{
				PeakData->f32AziAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32EleAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32Power[i][k][0] = 0.0f;
			}

			PeakData->s16AngCnt[i][k] = ang_cnt;

			/* for Debug */
#ifdef WIN32
			for (ele_idx = 0; ele_idx < ele_num_aps_step; ele_idx++)
			{
				for (azi_idx = 0; azi_idx < azi_num_aps_step; azi_idx++)
				{
					PeakData->f32APS[i][ele_idx * azi_num_aps_step + azi_idx] = temp_aps_mr[i * num_azi_ele + ele_idx * azi_num_aps_step + azi_idx];
				}
			}
#endif
		}
	}

}

#ifdef CUDA_ACC
static void Obj_vAngleEstimation_SR(Cuda_PeakData_t *PeakData, MultiMod_t CurMod)
#else
static void Obj_vAngleEstimation_SR(PeakData_t *PeakData, MultiMod_t CurMod)
#endif 
{
#ifdef LINUX
	struct timeval startTime, endTime;
	double diffTime;
#endif  

	uint16_t i, azi_idx, ele_idx, rx, tx, ch, j, ii, k;


	float32_t temp_abs_sum;
	float32_t aps_sll_th[OBJ_NUM_PEAK_HARM_MAX][OBJ_CO_NUM_HARMONICS_MAX];

#ifndef CUDA_ACC
	cmplxf32_t ch_fft[OBJ_NUM_PEAK_MAX][OBJ_CO_NUM_HARMONICS_MAX][OBJ_AE_NUM_CH_PARALLE];
	cmplxf32_t cmplx_sum;
	float32_t temp_aps[OBJ_NUM_PEAK_MAX][OBJ_CO_NUM_HARMONICS_MAX][OBJ_AE_ELE_NUM_APS_STEPS_MAX][OBJ_AE_AZI_NUM_APS_STEPS_MAX];
#endif

	float32_t num_azi, den_azi, dazi;
	float32_t num_ele, den_ele, dele;
	float32_t ch_norm_factor;

	float32_t Peak_Candi[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_azi_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_ele_idx[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];
	int16_t Peak_near_flg[OBJ_AE_CANDI_NUM_TARGET_MAX];

	int16_t peak_cnt;
	int16_t ang_cnt;

	float32_t temp_peak;
	int16_t temp_azi_idx;
	int16_t temp_ele_idx;
	int16_t temp_peak_azi_idx;
	int16_t temp_peak_ele_idx;
	int16_t temp_peak_flg;


	cmplxf32_t temp_fft[OBJ_AE_NUM_CH_PARALLE];

	ch_norm_factor = _sp_pow2(1.0f / OBJ_AE_NUM_CH);

	int16_t azi_num_aps_step;
	int16_t ele_num_aps_step;
	float32_t sll_th;
	float32_t aps_multi_th;
	float32_t azi_aps_step_size;
	float32_t ele_aps_step_size;
	float32_t chirp_interval;
	float32_t tdm_comp_constant;
	uint32_t temp_index;
	uint32_t pre_temp_index;

	int16_t num_azi_ele;

	float32_t azi_offset, ele_offset;
	float32_t azi_diff, ele_diff;

	int16_t temp_peak_cnt;
	float32_t peak_azi_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];
	float32_t peak_ele_temp_idx[OBJ_AE_NUM_PEAK_TARGET_MAX];

	azi_num_aps_step = OBJ_AE_AZI_SR_NUM_APS_STEPS;
	ele_num_aps_step = OBJ_AE_ELE_SR_NUM_APS_STEPS;
	sll_th = OBJ_AE_SR_APS_SLL_TH;
	aps_multi_th = OBJ_AE_SR_MULTI_TARGET_TH;
	azi_aps_step_size = OBJ_AE_AZI_SR_APS_STEP_SIZE;
	ele_aps_step_size = OBJ_AE_ELE_SR_APS_STEP_SIZE;
	chirp_interval = SP_SR_CHIRP_INTERVAL;

	num_azi_ele = azi_num_aps_step * ele_num_aps_step;

#ifdef LINUX
	gettimeofday(&startTime, NULL);
#endif

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{

		for (k = 0; k < 1; k++)
		{

#if (OBJ_HEXA_ARRAY_ON==1)

#ifdef CUDA_ACC
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], (cmplxf32_t*)ch_fft_sr + i * OBJ_AE_NUM_CH);
#else
			Obj_vAryRemov(PeakData->cf32PhFFTData[i], ch_fft[i][k]);
#endif

#else

#ifdef CUDA_ACC
			Obj_vAryPaste(temp_fft[i][k], (cmplxf32_t*)ch_fft + i * OBJ_AE_NUM_CH);
#else
			Obj_vAryPaste(temp_fft[i][k], ch_fft[i][k]);
#endif

#endif
			
			if (PeakData->rdu16phFreqIdx[i].D == 0)
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i] * 0.5f);
			}
			else
			{
				temp_abs_sum = _sp_pow2(PeakData->f32PhSllPow[i]);
			}

			aps_sll_th[i][k] = (temp_abs_sum * (float32_t)OBJ_AE_NUM_CH) * sll_th * ch_norm_factor;
		}
	}

	hipComplex alpha;
	hipComplex beta;
	alpha.x = 1.0; alpha.y = 0.0;
	beta.x = 0.0; beta.y = 0.0;

	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
			hipblasCgemv(cublas_handler, HIPBLAS_OP_T, OBJ_AE_NUM_CH, num_azi_ele, &alpha, (hipComplex*)PeakData->cf32SteerVec, OBJ_AE_NUM_CH, \
				ch_fft_sr + OBJ_AE_NUM_CH * i, 1, &beta, cmplx_sum_sr + num_azi_ele * i, 1);
		}
	}


#ifdef CUDA_ACC
	cal_temp_aps << < (uint32_t)(PeakData->s16PhCnt *num_azi_ele / MAX_NUM_OF_GPU_THREAD) + 1, MAX_NUM_OF_GPU_THREAD >> > (temp_aps_sr, cmplx_sum_sr, ch_norm_factor);
#ifdef LINUX
	(hipStreamAttachMemAsync(NULL, temp_aps_sr, 0, hipMemAttachHost));
#else
	(hipStreamAttachMemAsync(NULL, temp_aps_sr, 0, hipMemAttachGlobal));
#endif
	(hipStreamSynchronize(NULL));
#endif


	for (i = 0; i < PeakData->s16PhCnt; i++)
	{
		for (k = 0; k < 1; k++)
		{
			peak_cnt = 0;
			temp_peak_azi_idx = 0;
			temp_peak_ele_idx = 0;
			for (ele_idx = 1; ele_idx < ele_num_aps_step - 1; ele_idx++)
			{
				_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);
				for (azi_idx = 1; azi_idx < azi_num_aps_step - 1; azi_idx++)
				{
					_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);
					temp_index = i * num_azi_ele + ele_idx * azi_num_aps_step;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
					if ((temp_aps_sr[temp_index + azi_idx] > aps_sll_th[i][k]))
#else
					if ((temp_aps_sr[temp_index + azi_idx] > aps_sll_th[i][k])
						&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_idx - 1])
						&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_idx + 1])
						&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index - azi_num_aps_step + azi_idx])
						&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_num_aps_step + azi_idx]))
#endif
					{
						_sp_cnt_break(peak_cnt, OBJ_AE_CANDI_NUM_TARGET_MAX);

						Peak_Candi[peak_cnt] = temp_aps_sr[temp_index + azi_idx];
						Peak_azi_idx[peak_cnt] = azi_idx;
						Peak_ele_idx[peak_cnt] = ele_idx;

#if (OBJ_AE_ADD_OBJ_ENABLE == 1)
						if ((temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_idx - 1])
							&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_idx + 1])
							&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index - azi_num_aps_step + azi_idx])
							&& (temp_aps_sr[temp_index + azi_idx] > temp_aps_sr[temp_index + azi_num_aps_step + azi_idx]))
						{
							Peak_flg[peak_cnt] = 1U;
						}
						else
						{
							Peak_flg[peak_cnt] = 0U;
						}
#else
						Peak_flg[peak_cnt] = 1U;
#endif
						peak_cnt++;
					}
				}
			}

			for (ii = 0; ii < peak_cnt; ++ii)
			{
				for (j = ii + 1; j < peak_cnt; ++j)
				{

					if (Peak_Candi[ii] < Peak_Candi[j])
					{
						temp_peak = Peak_Candi[ii];
						Peak_Candi[ii] = Peak_Candi[j];
						Peak_Candi[j] = temp_peak;

						temp_azi_idx = Peak_azi_idx[ii];
						Peak_azi_idx[ii] = Peak_azi_idx[j];
						Peak_azi_idx[j] = temp_azi_idx;

						temp_ele_idx = Peak_ele_idx[ii];
						Peak_ele_idx[ii] = Peak_ele_idx[j];
						Peak_ele_idx[j] = temp_ele_idx;

						temp_peak_flg = Peak_flg[ii];
						Peak_flg[ii] = Peak_flg[j];
						Peak_flg[j] = temp_peak_flg;
					}
				}
			}


			temp_peak_cnt = 0;
			for (j = 0; j < peak_cnt; j++)
			{
				if (Peak_flg[j] == 1)
				{
					_sp_cnt_break(temp_peak_cnt, OBJ_AE_NUM_PEAK_TARGET_MAX);
					peak_azi_temp_idx[temp_peak_cnt] = Peak_azi_idx[j];
					peak_ele_temp_idx[temp_peak_cnt] = Peak_ele_idx[j];
					temp_peak_cnt++;
				}
			}

			for (j = 0; j < peak_cnt; j++)
			{
				for (ii = 0; ii < temp_peak_cnt; ii++)
				{
					azi_diff = _sp_pow2(_sp_abs(peak_azi_temp_idx[ii] - Peak_azi_idx[j]));
					ele_diff = _sp_pow2(_sp_abs(peak_ele_temp_idx[ii] - Peak_ele_idx[j]));

					if (_sp_sqrtf(azi_diff + ele_diff) < 4.0f)
					{
						Peak_near_flg[j] = 1;
					}
				}
			}

			ang_cnt = 0;
			PeakData->s16AngCnt[i][k] = 0;
			if (peak_cnt > 0)
			{
				for (j = 0; j < peak_cnt; j++)
				{
					if ((Peak_Candi[j] > aps_multi_th*Peak_Candi[0])
						&& (Peak_near_flg[j] == 1))
					{

						_sp_cnt_break(ang_cnt, OBJ_AE_NUM_TARGET_MAX);
						temp_index = i * num_azi_ele;

						num_azi = temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];
						den_azi = temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + 1] - (2.0f * temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - 1];

						num_ele = temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];
						den_ele = temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] + azi_num_aps_step] - (2.0f * temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]) + temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j] - azi_num_aps_step];

						dazi = 0.5f * azi_aps_step_size;
						dele = 0.5f * ele_aps_step_size;

						azi_offset = (num_azi / den_azi) * dazi;
						ele_offset = (num_ele / den_ele) * dele;

						if (_sp_abs(azi_offset) < 1.0f)
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]] + azi_offset;
						}
						else
						{
							PeakData->f32AziAngle[i][k][ang_cnt] = PeakData->f32AziAngTable[Peak_azi_idx[j]];
						}

						if (_sp_abs(ele_offset) < 2.0f)
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]] + ele_offset;
						}
						else
						{
							PeakData->f32EleAngle[i][k][ang_cnt] = PeakData->f32EleAngTable[Peak_ele_idx[j]];
						}

						PeakData->f32Power[i][k][ang_cnt] = 10.0f*_sp_log10f(temp_aps_sr[temp_index + Peak_ele_idx[j] * azi_num_aps_step + Peak_azi_idx[j]]);

						if (Peak_flg[j] == 1U)
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 1;
						}
						else
						{
							PeakData->s16TrackFlag[i][k][ang_cnt] = 0;
						}

						ang_cnt++;
					}
				}
			}
			else
			{
				PeakData->f32AziAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32EleAngle[i][k][0] = OBJ_AE_FAILURE_ANGLE;
				PeakData->f32Power[i][k][0] = 0.0f;
			}

			PeakData->s16AngCnt[i][k] = ang_cnt;

			/* for Debug */
#ifdef WIN32
			for (ele_idx = 0; ele_idx < ele_num_aps_step; ele_idx++)
			{
				for (azi_idx = 0; azi_idx < azi_num_aps_step; azi_idx++)
				{
					PeakData->f32APS[i][ele_idx * azi_num_aps_step + azi_idx] = temp_aps_sr[i * num_azi_ele + ele_idx * azi_num_aps_step + azi_idx];
				}
			}
#endif
		}
	}

}


static cmplxf32_t Obj_cf32GetPhaseDiff(const cmplxf32_t a, const cmplxf32_t b)
{
	cmplxf32_t c;
	float32_t temp_mag;

	c.real = (a.real * b.real) + (a.imag * b.imag);
	c.imag = (a.imag * b.real) - (a.real * b.imag);
	temp_mag = _sp_sqrtf((c.real * c.real) + (c.imag * c.imag));

	c.real = c.real / temp_mag;
	c.imag = c.imag / temp_mag;

	return c;
}

static void Obj_vAryRemov(cmplxf32_t temp_peak_fft[], cmplxf32_t temp_ch_fft[])
{
	int16_t remov_ch_idx[OBJ_AE_NUM_CH_REMOV] = OBJ_AE_CH_REMOV_IDX;
	int16_t temp_idx;
	int16_t ch;
	int16_t cnt = 0;
	int16_t cnt_rmv = 0;

	for (ch = 0; ch < OBJ_AE_NUM_CH_PARALLE; ch++)
	{
		temp_idx = remov_ch_idx[cnt_rmv];
		if (temp_idx == ch)
		{
			cnt_rmv += 1;
		}
		else
		{
			temp_ch_fft[cnt].real = temp_peak_fft[ch].real;
			temp_ch_fft[cnt].imag = temp_peak_fft[ch].imag;
			cnt += 1;
		}
	}
}

static void Obj_vAryPaste(cmplxf32_t temp_peak_fft[], cmplxf32_t temp_ch_fft[])
{
	int16_t ch;

	for (ch = 0; ch < OBJ_AE_NUM_CH_PARALLE; ch++)
	{
		temp_ch_fft[ch].real = temp_peak_fft[ch].real;
		temp_ch_fft[ch].imag = temp_peak_fft[ch].imag;
	}
}

void alloc_peak_data_gpu_mem(void)
{
#ifdef CUDA_ACC
	size_t sizeOfcf32SteerVec_lr = OBJ_AE_ELE_LR_NUM_APS_STEPS * OBJ_AE_AZI_LR_NUM_APS_STEPS * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);
	size_t sizeOfcf32SteerVec_mr = OBJ_AE_ELE_MR_NUM_APS_STEPS * OBJ_AE_AZI_MR_NUM_APS_STEPS  * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);
	size_t sizeOfcf32SteerVec_sr = OBJ_AE_ELE_SR_NUM_APS_STEPS * OBJ_AE_AZI_SR_NUM_APS_STEPS  * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);

	(hipMallocManaged((void**)&(gLR_PeakData.cf32SteerVec), sizeOfcf32SteerVec_lr));
	(hipMallocManaged((void**)&(gMR_PeakData.cf32SteerVec), sizeOfcf32SteerVec_mr));
	(hipMallocManaged((void**)&(gSR_PeakData.cf32SteerVec), sizeOfcf32SteerVec_sr));


	/*
	size_t sizeOfch_fft = OBJ_NUM_CANDIDATE_OBJ * OBJ_CO_NUM_HARMONICS_MAX * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);
	(hipMallocManaged((void**)&ch_fft, sizeOfch_fft));

	size_t sizeOfcmplx_sum = (OBJ_CO_NUM_HARMONICS_MAX * OBJ_NUM_CANDIDATE_OBJ * OBJ_AE_AZI_NUM_APS_STEPS_MAX * OBJ_AE_ELE_NUM_APS_STEPS_MAX / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(cmplxf32_t);
	(hipMallocManaged((void**)&cmplx_sum, sizeOfcmplx_sum));

	size_t sizeOftemp_aps = (OBJ_CO_NUM_HARMONICS_MAX * OBJ_NUM_CANDIDATE_OBJ * OBJ_AE_AZI_NUM_APS_STEPS_MAX * OBJ_AE_ELE_NUM_APS_STEPS_MAX / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(float32_t);
	(hipMallocManaged((void**)&temp_aps, sizeOftemp_aps));
	*/

	size_t sizeOfch_fft_lr = OBJ_NUM_PEAK_HARM_LR * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);
	size_t sizeOfch_fft_mr = OBJ_NUM_PEAK_HARM_MR * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);
	size_t sizeOfch_fft_sr = OBJ_NUM_PEAK_HARM_SR * OBJ_AE_NUM_CH * sizeof(cmplxf32_t);

	(hipMallocManaged((void**)&ch_fft_lr, sizeOfch_fft_lr));
	(hipMallocManaged((void**)&ch_fft_mr, sizeOfch_fft_mr));
	(hipMallocManaged((void**)&ch_fft_sr, sizeOfch_fft_sr));

	size_t sizeOfcmplx_sum_lr = (OBJ_NUM_PEAK_HARM_LR * OBJ_AE_AZI_LR_NUM_APS_STEPS * OBJ_AE_ELE_LR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(cmplxf32_t);
	size_t sizeOfcmplx_sum_mr = (OBJ_NUM_PEAK_HARM_MR * OBJ_AE_AZI_MR_NUM_APS_STEPS * OBJ_AE_ELE_MR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(cmplxf32_t);
	size_t sizeOfcmplx_sum_sr = (OBJ_NUM_PEAK_HARM_SR * OBJ_AE_AZI_SR_NUM_APS_STEPS * OBJ_AE_ELE_SR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(cmplxf32_t);

	(hipMallocManaged((void**)&cmplx_sum_lr, sizeOfcmplx_sum_lr));
	(hipMallocManaged((void**)&cmplx_sum_mr, sizeOfcmplx_sum_mr));
	(hipMallocManaged((void**)&cmplx_sum_sr, sizeOfcmplx_sum_sr));


	size_t sizeOftemp_aps_lr = (OBJ_NUM_PEAK_HARM_LR * OBJ_AE_AZI_LR_NUM_APS_STEPS * OBJ_AE_ELE_LR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(float32_t);
	size_t sizeOftemp_aps_mr = (OBJ_NUM_PEAK_HARM_MR * OBJ_AE_AZI_MR_NUM_APS_STEPS * OBJ_AE_ELE_MR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(float32_t);
	size_t sizeOftemp_aps_sr = (OBJ_NUM_PEAK_HARM_SR * OBJ_AE_AZI_SR_NUM_APS_STEPS * OBJ_AE_ELE_SR_NUM_APS_STEPS / MAX_NUM_OF_GPU_THREAD + 1) * MAX_NUM_OF_GPU_THREAD * sizeof(float32_t);

	(hipMallocManaged((void**)&temp_aps_lr, sizeOftemp_aps_lr));
	(hipMallocManaged((void**)&temp_aps_mr, sizeOftemp_aps_mr));
	(hipMallocManaged((void**)&temp_aps_sr, sizeOftemp_aps_sr));


	hipblasCreate(&cublas_handler);
#endif
}


