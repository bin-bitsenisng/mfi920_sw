
#include <hip/hip_runtime.h>
#ifdef CUDA_ACC

#ifdef WIN32
extern "C" {
#include "Cuda_bts_lib.cuh"
}
#else
#include "Cuda_bts_lib.cuh"
#endif
#include "device_launch_parameters.h"
#include <iostream>
#include <chrono>
#include <cufft.h>
#include <cublas_v2.h>
#include <cuda_runtime_api.h>
#include <cuda_runtime.h>
using namespace std;
void Init_cuda_for_rm(void);
void Init_cuda_for_lms_mode(void);
void windowing_2d(uint32_t num_tx, uint32_t num_rx, uint32_t num_chirps, uint32_t num_samples, int16_t * input, cufftReal * output, float32_t * wincoeff);

cublasHandle_t cublas_handle;

// direct fft2d 
cufftHandle rm_plan_direct_2d_fft;
cufftHandle plan_direct_2d_fft;

cufftReal* lr_windowed_1d_data;
cufftReal* mr_windowed_1d_data;
cufftReal* sr_windowed_1d_data;
cufftReal* rm_windowed_1d_data;

extern float32_t Sim_PowSpectrum[OBJ_NUM_FFT_R][OBJ_NUM_FFT_D];

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(cudaError_t code, const char* file, int line, bool abort = true)
{
	if (code != cudaSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
		if (abort) { getchar(); exit(code); }
	}
}

/*********************/
/* CUFFT ERROR CHECK */
/*********************/
static const char* _cudaGetErrorEnum(cufftResult error)
{
	switch (error)
	{
	case CUFFT_SUCCESS:
		return "CUFFT_SUCCESS";

	case CUFFT_INVALID_PLAN:
		return "CUFFT_INVALID_PLAN";

	case CUFFT_ALLOC_FAILED:
		return "CUFFT_ALLOC_FAILED";

	case CUFFT_INVALID_TYPE:
		return "CUFFT_INVALID_TYPE";

	case CUFFT_INVALID_VALUE:
		return "CUFFT_INVALID_VALUE";

	case CUFFT_INTERNAL_ERROR:
		return "CUFFT_INTERNAL_ERROR";

	case CUFFT_EXEC_FAILED:
		return "CUFFT_EXEC_FAILED";

	case CUFFT_SETUP_FAILED:
		return "CUFFT_SETUP_FAILED";

	case CUFFT_INVALID_SIZE:
		return "CUFFT_INVALID_SIZE";

	case CUFFT_UNALIGNED_DATA:
		return "CUFFT_UNALIGNED_DATA";
	}

	return "<unknown>";
}

#define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
inline void __cufftSafeCall(cufftResult err, const char* file, const int line)
{
	if (CUFFT_SUCCESS != err) {
		fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n", __FILE__, __LINE__, err, \
			_cudaGetErrorEnum(err)); \
			cudaDeviceReset(); assert(0); \
	}
}

/**********************/
/* CUBLAS ERROR CHECK */
/**********************/
static const char* _cudaCublasGetErrorEnum(cublasStatus_t error)
{
	switch (error)
	{
	case CUBLAS_STATUS_SUCCESS:
		return "CUBLAS_STATUS_SUCCESS";

	case CUBLAS_STATUS_NOT_INITIALIZED:
		return "CUBLAS_STATUS_NOT_INITIALIZED";

	case CUBLAS_STATUS_ALLOC_FAILED:
		return "CUBLAS_STATUS_ALLOC_FAILED";

	case CUBLAS_STATUS_INVALID_VALUE:
		return "CUBLAS_STATUS_INVALID_VALUE";

	case CUBLAS_STATUS_ARCH_MISMATCH:
		return "CUBLAS_STATUS_ARCH_MISMATCH";

	case CUBLAS_STATUS_MAPPING_ERROR:
		return "CUBLAS_STATUS_MAPPING_ERROR";

	case CUBLAS_STATUS_EXECUTION_FAILED:
		return "CUBLAS_STATUS_EXECUTION_FAILED";

	case CUBLAS_STATUS_INTERNAL_ERROR:
		return "CUBLAS_STATUS_INTERNAL_ERROR";

	case CUBLAS_STATUS_NOT_SUPPORTED:
		return "CUBLAS_STATUS_NOT_SUPPORTED";

	case CUBLAS_STATUS_LICENSE_ERROR:
		return "CUBLAS_STATUS_LICENSE_ERROR";
	}

	return "<unknown>";
}

#define cublasSafeCall(err)      __cublasSafeCall(err, __FILE__, __LINE__)
inline void __cublasSafeCall(cublasStatus_t err, const char* file, const int line)
{
	if (CUBLAS_STATUS_SUCCESS != err) {
		fprintf(stderr, "Cublas error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n", __FILE__, __LINE__, err, \
			_cudaCublasGetErrorEnum(err)); \
			cudaDeviceReset(); assert(0); \
	}
}

__global__ void cal_mag2D(cmplxf32_t* src, float* dst, int src_column_size)
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int src_index = gridDim.x * src_column_size*(blockIdx.y * gridDim.z + blockIdx.z) + (bi * src_column_size) + ti;
	int dst_index = gridDim.x*blockDim.x*(blockIdx.y*gridDim.z + blockIdx.z) + bi + ti * gridDim.x;
	dst[dst_index] = (src[src_index].real) * (src[src_index].real) + (src[src_index].imag) * (src[src_index].imag);
}
__global__ void sum_mag(int txrx_size, int offset, float* src, float* dst)
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int bd = blockDim.x * (bi + offset);
	int arr_size = blockDim.x * (gridDim.x + offset);
	int index = bd + ti;

	for (int i = 0; i < txrx_size; i++)
	{
		dst[index] += src[arr_size * i + index];
	}

	dst[index] = sqrt(dst[index]);
}

__global__ void cal_nci(cmplxf32_t* src, float* dst, int offset, int num_tx, int num_rx) // src : cf32fftData_2D, dst : chrips x samples
{
	int ti = threadIdx.x;
	int ci = blockIdx.x;
	int dst_index = 0;
	int src_index = 0;
	dst_index = blockDim.x*ci + ti;
	for (int txrx = 0; txrx < num_tx*num_rx; txrx++) {
		src_index = (gridDim.x * blockDim.x)*(txrx)+ci * blockDim.x + ti;
		dst[dst_index] += (src[src_index].real) * (src[src_index].real) + (src[src_index].imag) * (src[src_index].imag);
	}
	// sqrt
	dst[dst_index] = sqrt(dst[dst_index]);
}
__global__ void transpose_nci(float* src, float* dst, int offset)
{
	int ti = threadIdx.x;
	int ci = blockIdx.x + offset;
	int dst_index = blockDim.x*ci + ti;
	int src_index = ti * (gridDim.x + offset + 1) + ci;
	dst[dst_index] = src[src_index];
}
__global__ void window_2d_coeff_mul(int16_t* src, cufftReal* dst, float32_t* coeff)
{
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int index = bi * blockDim.x + ti;

	dst[index] = (float32_t)(src[index]) * coeff[index];
}

void windowing_2d(uint32_t num_tx, uint32_t num_rx, uint32_t num_chirps, uint32_t num_samples, int16_t* input, cufftReal* output, float32_t* wincoeff)
{
	for (int tx = 0; tx < num_tx; tx++)
	{
		for (int rx = 0; rx < num_rx; rx++)
		{
			window_2d_coeff_mul << <num_chirps, num_samples >> > (input + tx * num_rx * num_samples * num_chirps + rx * num_samples * num_chirps, \
				output + tx * num_rx * num_samples * num_chirps + rx * num_samples * num_chirps, \
				wincoeff);
		}
	}
}
#ifdef WIN32
extern "C"
#endif
int fft_2d_cuda(void)
{
	// 2D Windowing
	windowing_2d(SP_NUM_TX, SP_NUM_RX, SP_NUM_CHIRPS, SP_NUM_SAMPLES, gLR_PreData.s16adcData, lr_windowed_1d_data, gLR_PreWindow.cf32WinCoeffMap);
	windowing_2d(SP_NUM_TX, SP_NUM_RX, SP_NUM_CHIRPS, SP_NUM_SAMPLES, gMR_PreData.s16adcData, mr_windowed_1d_data, gMR_PreWindow.cf32WinCoeffMap);
	windowing_2d(SP_NUM_TX, SP_NUM_RX, SP_NUM_CHIRPS, SP_NUM_SAMPLES, gSR_PreData.s16adcData, sr_windowed_1d_data, gSR_PreWindow.cf32WinCoeffMap);
	windowing_2d(SP_RM_NUM_TX, SP_NUM_RX, SP_RM_NUM_CHIRPS, SP_RM_NUM_SAMPLES, gRM_PreData.s16adcData, rm_windowed_1d_data, gRM_PreWindow.cf32WinCoeffMap);

	gpuErrchk(cudaStreamSynchronize(NULL));

	// 2D FFT
	int src_index = 0;
	int dst_index = 0;
	for (int tx = 0; tx < SP_RM_NUM_TX; tx++) {
		for (int rx = 0; rx < SP_NUM_RX; rx++) {
			src_index = SP_RM_NUM_SAMPLES * SP_RM_NUM_CHIRPS*(tx * SP_NUM_RX + rx);
			dst_index = (SP_RM_NUM_SAMPLES *0.5 + 1) * SP_RM_NUM_CHIRPS*(tx * SP_NUM_RX + rx);
			cufftSafeCall(cufftExecR2C(rm_plan_direct_2d_fft, (cufftReal*)(rm_windowed_1d_data + src_index), (cufftComplex*)(gRM_PreData.cf32fftData_2D + dst_index)));
		}
	}
	for (int tx = 0; tx < SP_NUM_TX; tx++) {
		for (int rx = 0; rx < SP_NUM_RX; rx++) {
			src_index = SP_NUM_SAMPLES * SP_NUM_CHIRPS*(tx * SP_NUM_RX + rx);
			dst_index = (SP_NUM_SAMPLES*0.5 + 1)*SP_NUM_CHIRPS*(tx * SP_NUM_RX + rx);
			cufftSafeCall(cufftExecR2C(plan_direct_2d_fft, (cufftReal*)(lr_windowed_1d_data + src_index), (cufftComplex*)(gLR_PreData.cf32fftData_2D + dst_index)));
			cufftSafeCall(cufftExecR2C(plan_direct_2d_fft, (cufftReal*)(mr_windowed_1d_data + src_index), (cufftComplex*)(gMR_PreData.cf32fftData_2D + dst_index)));
			cufftSafeCall(cufftExecR2C(plan_direct_2d_fft, (cufftReal*)(sr_windowed_1d_data + src_index), (cufftComplex*)(gSR_PreData.cf32fftData_2D + dst_index)));
		}
	}

#ifdef LINUX
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gLR_PreData.cf32fftData_2D, 0, cudaMemAttachHost));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gMR_PreData.cf32fftData_2D, 0, cudaMemAttachHost));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gSR_PreData.cf32fftData_2D, 0, cudaMemAttachHost));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gRM_PreData.cf32fftData_2D, 0, cudaMemAttachHost));
#else
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gLR_PreData.cf32fftData_2D, 0, cudaMemAttachGlobal));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gMR_PreData.cf32fftData_2D, 0, cudaMemAttachGlobal));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gSR_PreData.cf32fftData_2D, 0, cudaMemAttachGlobal));
	gpuErrchk(cudaStreamAttachMemAsync(NULL, gRM_PreData.cf32fftData_2D, 0, cudaMemAttachGlobal));
#endif
	gpuErrchk(cudaStreamSynchronize(NULL));

	return 0;
}


#ifdef WIN32
extern "C"
#endif
void Init_cuda(void)
{
	Init_cuda_for_rm();
	Init_cuda_for_lms_mode();

	cublasSafeCall(cublasCreate(&cublas_handle));
}

void Init_cuda_for_rm(void)
{
	// Reference Mode
	size_t rm_sizeOfs16adcData = SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX * SP_RM_NUM_SAMPLES * sizeof(int16_t);
	size_t rm_sizeOfWindowed1dData = SP_RM_NUM_CHIRPS * SP_RM_NUM_TX * SP_NUM_RX * SP_RM_NUM_FFT_R * sizeof(cmplxf32_t);
	size_t rm_sizeOfcf32fftData_2D = (SP_RM_NUM_FFT_R *0.5 + 1) * SP_RM_NUM_TX * SP_NUM_RX * SP_RM_NUM_FFT_D * sizeof(cmplxf32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gRM_PreData.s16adcData), rm_sizeOfs16adcData));
	gpuErrchk(cudaMallocManaged((void**)&rm_windowed_1d_data, rm_sizeOfWindowed1dData));
	gpuErrchk(cudaMallocManaged((void**)&(gRM_PreData.cf32fftData_2D), rm_sizeOfcf32fftData_2D));
	size_t rm_sizeOfcf32WinCoeffMap = SP_RM_NUM_FFT_R * SP_RM_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gRM_PreWindow.cf32WinCoeffMap), rm_sizeOfcf32WinCoeffMap));
	size_t rm_sizeOfgPhaseCal = SP_RM_NUM_TX * SP_NUM_RX * sizeof(cmplxf32_t);
	gpuErrchk(cudaMallocManaged((void**)&gPhaseCal.f32RM_PhaseCal, rm_sizeOfgPhaseCal));
	size_t rm_sizeOff32PowSpectrum = SP_RM_NUM_FFT_R_HALF * SP_RM_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gRM_BeamData.f32PowSpectrum), rm_sizeOff32PowSpectrum));
	size_t rm_sizeOff32PowSpectrum_temp = (SP_RM_NUM_FFT_R_HALF + 1) * SP_RM_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gRM_BeamData.f32PowSpectrum_temp), rm_sizeOff32PowSpectrum_temp));

	cufftSafeCall(cufftPlan2d(&rm_plan_direct_2d_fft, SP_RM_NUM_FFT_D, SP_RM_NUM_SAMPLES, CUFFT_R2C));
}

void Init_cuda_for_lms_mode(void)
{
	// LMS Mode
	size_t sizeOfs16adcData = SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX * SP_NUM_SAMPLES * sizeof(int16_t);
	size_t sizeOfWindowed1dData = SP_NUM_CHIRPS * SP_NUM_TX * SP_NUM_RX * SP_NUM_FFT_R * sizeof(cufftReal);
	size_t sizeOfcf32fftData_2D = (SP_NUM_FFT_R*0.5 + 1) * SP_NUM_TX * SP_NUM_RX * SP_NUM_FFT_D * sizeof(cmplxf32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gLR_PreData.s16adcData), sizeOfs16adcData));
	gpuErrchk(cudaMallocManaged((void**)&(gMR_PreData.s16adcData), sizeOfs16adcData));
	gpuErrchk(cudaMallocManaged((void**)&(gSR_PreData.s16adcData), sizeOfs16adcData));
	gpuErrchk(cudaMallocManaged((void**)&lr_windowed_1d_data, sizeOfWindowed1dData));
	gpuErrchk(cudaMallocManaged((void**)&mr_windowed_1d_data, sizeOfWindowed1dData));
	gpuErrchk(cudaMallocManaged((void**)&sr_windowed_1d_data, sizeOfWindowed1dData));
	gpuErrchk(cudaMallocManaged((void**)&(gLR_PreData.cf32fftData_2D), sizeOfcf32fftData_2D));
	gpuErrchk(cudaMallocManaged((void**)&(gMR_PreData.cf32fftData_2D), sizeOfcf32fftData_2D));
	gpuErrchk(cudaMallocManaged((void**)&(gSR_PreData.cf32fftData_2D), sizeOfcf32fftData_2D));

	size_t sizeOfcf32WinCoeffMap = SP_NUM_FFT_R * SP_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gLR_PreWindow.cf32WinCoeffMap), sizeOfcf32WinCoeffMap));
	gpuErrchk(cudaMallocManaged((void**)&(gMR_PreWindow.cf32WinCoeffMap), sizeOfcf32WinCoeffMap));
	gpuErrchk(cudaMallocManaged((void**)&(gSR_PreWindow.cf32WinCoeffMap), sizeOfcf32WinCoeffMap));

	size_t sizeOfgPhaseCal = SP_NUM_TX * SP_NUM_RX * sizeof(cmplxf32_t);
	gpuErrchk(cudaMallocManaged((void**)&gPhaseCal.f32LR_PhaseCal, sizeOfgPhaseCal));
	gpuErrchk(cudaMallocManaged((void**)&gPhaseCal.f32MR_PhaseCal, sizeOfgPhaseCal));
	gpuErrchk(cudaMallocManaged((void**)&gPhaseCal.f32SR_PhaseCal, sizeOfgPhaseCal));

	size_t sizeOff32PowSpectrum_temp = (SP_NUM_FFT_R_HALF + 1) * SP_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gLR_BeamData.f32PowSpectrum_temp), sizeOff32PowSpectrum_temp));
	gpuErrchk(cudaMallocManaged((void**)&(gMR_BeamData.f32PowSpectrum_temp), sizeOff32PowSpectrum_temp));
	gpuErrchk(cudaMallocManaged((void**)&(gSR_BeamData.f32PowSpectrum_temp), sizeOff32PowSpectrum_temp));

	size_t sizeOff32PowSpectrum = SP_NUM_FFT_R_HALF * SP_NUM_FFT_D * sizeof(float32_t);
	gpuErrchk(cudaMallocManaged((void**)&(gLR_BeamData.f32PowSpectrum), sizeOff32PowSpectrum));
	gpuErrchk(cudaMallocManaged((void**)&(gMR_BeamData.f32PowSpectrum), sizeOff32PowSpectrum));
	gpuErrchk(cudaMallocManaged((void**)&(gSR_BeamData.f32PowSpectrum), sizeOff32PowSpectrum));

	cufftSafeCall(cufftPlan2d(&plan_direct_2d_fft, SP_NUM_FFT_D, SP_NUM_SAMPLES, CUFFT_R2C));
}

#ifdef WIN32
extern "C"
#endif
void Obj_vNCI_cuda(uint32_t num_tx, uint32_t num_rx, uint32_t num_fft_r_half, uint32_t num_fft_d, uint32_t nci_range_init_idx, Cuda_PreData_t * PreData, Cuda_BeamData_t * BeamData)
{
	int offset = nci_range_init_idx;
	cal_nci << <num_fft_r_half + 1, num_fft_d >> > (PreData->cf32fftData_2D, BeamData->f32PowSpectrum_temp, offset, num_tx, num_rx);	
	transpose_nci << <num_fft_r_half - offset, num_fft_d >> > (BeamData->f32PowSpectrum_temp, BeamData->f32PowSpectrum, offset);
#ifdef LINUX
	gpuErrchk(cudaStreamAttachMemAsync(NULL, BeamData->f32PowSpectrum, 0, cudaMemAttachHost));
#else
	gpuErrchk(cudaStreamAttachMemAsync(NULL, BeamData->f32PowSpectrum, 0, cudaMemAttachGlobal));
#endif
	gpuErrchk(cudaStreamSynchronize(NULL));

}

#endif